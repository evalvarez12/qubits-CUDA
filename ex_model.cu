#include "hip/hip_runtime.h"
#ifndef EXMODEL
# define EXMODEL

namespace extra_model{

void modelVar1(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(3,14,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar2(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,7,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar3(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(1,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,14,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar4(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,13,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar5(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,14,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar6(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(1,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,8,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,11,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar7(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,8,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,6,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar8(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,12,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar9(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(1,8,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,15,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar10(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,6,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar11(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,9,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar12(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,8,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,12,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar13(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,8,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,8,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,15,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelVar14(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);

  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelConexComplete(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion COMPLETA  A B  
  for(int i=0;i<xlen;i++) {
    for(int j=xlen;j<nqubits-1;j++) {
      Ui_kernel<<<numblocks,numthreads>>>(i,j,dev_R,dev_I,cos(jp),sin(jp),l);
    }
  }
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
void modelConexRand(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion VARIABLE  A B
  int num_conex=conA.size();
  for(int i=0;i<num_conex;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(conA(i),conB(i),dev_R,dev_I,cos(jp),sin(jp),l);
  }
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void modelConexRandB(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion A con B
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);  
  //la interaccion VARIABLE  0 A
  int num_conex=conA.size();
  for(int i=0;i<num_conex;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(conA(i),conB(i),dev_R,dev_I,cos(j),sin(j),l);
  }
  
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
  
void modelConexRandABC(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    } 
  //la interaccion VARIABLE  ABC
  int num_conex=conA.size();
  for(int i=0;i<num_conex;i++) {
    if (conA(i) == nqubits-1) {
      Ui_kernel<<<numblocks,numthreads>>>(conA(i),conB(i),dev_R,dev_I,cos(j),sin(j),l);
    }
    else {
      Ui_kernel<<<numblocks,numthreads>>>(conA(i),conB(i),dev_R,dev_I,cos(jp),sin(jp),l);
    }
  }
  
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
  
void model3_open_op1(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }  
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model3_open_op2(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(3,5,dev_R,dev_I,cos(js(1)),sin(js(1)),l);
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(7,11,dev_R,dev_I,cos(js(6)),sin(js(6)),l);
  Ui_kernel<<<numblocks,numthreads>>>(9,14,dev_R,dev_I,cos(js(9)),sin(js(9)),l);
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }   

void model3_open_op3(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(1,3,dev_R,dev_I,cos(js(1)),sin(js(1)),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,5,dev_R,dev_I,cos(js(3)),sin(js(3)),l);
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(6,8,dev_R,dev_I,cos(js(6)),sin(js(6)),l);
  Ui_kernel<<<numblocks,numthreads>>>(7,14,dev_R,dev_I,cos(js(7)),sin(js(7)),l);  
  Ui_kernel<<<numblocks,numthreads>>>(9,12,dev_R,dev_I,cos(js(8)),sin(js(8)),l);
  Ui_kernel<<<numblocks,numthreads>>>(12,15,dev_R,dev_I,cos(js(9)),sin(js(9)),l);
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model3_open_op4(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(2,4,dev_R,dev_I,cos(js(1)),sin(js(1)),l);
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(7,10,dev_R,dev_I,cos(js(6)),sin(js(6)),l);
  Ui_kernel<<<numblocks,numthreads>>>(10,14,dev_R,dev_I,cos(js(9)),sin(js(9)),l);
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model3_open_op5(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(0,2,dev_R,dev_I,cos(js(3)),sin(js(3)),l);    
  Ui_kernel<<<numblocks,numthreads>>>(2,4,dev_R,dev_I,cos(js(1)),sin(js(1)),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,5,dev_R,dev_I,cos(js(4)),sin(js(4)),l);
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(6,10,dev_R,dev_I,cos(js(6)),sin(js(6)),l);  
  Ui_kernel<<<numblocks,numthreads>>>(8,10,dev_R,dev_I,cos(js(7)),sin(js(7)),l);
  Ui_kernel<<<numblocks,numthreads>>>(10,13,dev_R,dev_I,cos(js(9)),sin(js(9)),l);
  Ui_kernel<<<numblocks,numthreads>>>(11,15,dev_R,dev_I,cos(js(12)),sin(js(12)),l);
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
 
void model3_open_op6(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  Ui_kernel<<<numblocks,numthreads>>>(0,2,dev_R,dev_I,cos(js(1)),sin(js(1)),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,4,dev_R,dev_I,cos(js(3)),sin(js(3)),l);
  Ui_kernel<<<numblocks,numthreads>>>(0,5,dev_R,dev_I,cos(js(4)),sin(js(4)),l);
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model3_open_VarMagnetic(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen,itpp::ivec conA, itpp::ivec conB){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  double kcosC,ksinC,bxC,byC,bzC;
  int l=pow(2,nqubits);
  
  double JP_real = .8;
  
  itpp::vec bC(3);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i+xlen)),sin(js(i+xlen)),l);
    }  
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(JP_real),sin(JP_real),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
    bC(0)=jp/std::sqrt(2.); bC(1)=0.; bC(2)=jp/std::sqrt(2.);
    set_parameters(bC,kcosC,ksinC,bxC,byC,bzC);
    Uk_kernel<<<numblocks,numthreads>>>(nqubits-1,dev_R,dev_I,bxC,byC,bzC,kcosC,ksinC,l);
  return;  
  }  
  

} 

#endif    