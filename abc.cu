#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <itpp/itbase.h>
#include <cpp/dev_random.cpp>
#include <cpp/itpp_ext_math.cpp>
#include <cpp/spinchain.cpp>
#include <math.h>
#include <tclap/CmdLine.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "tools.cpp"
#include "cuda_utils.cu"
#include "model.cu"
#include "ev_routines.cu"
#include "ev_math.cu"
#include "cuda_functions.cu"
#include "ex_model.cu"
#include <time.h>


TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<unsigned int> CseedArg("","Cseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<unsigned int> EseedArg("","Eseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<unsigned int> PARAMseedArg("","PARAMseed", "Random seed [0 for urandom]",false, 0,"unsigned int",cmd);
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"nichts", "string",cmd);
TCLAP::ValueArg<int> nqubitsArg("q","qubits", "Number of qubits",false, 3,"int",cmd);
TCLAP::ValueArg<int> numtArg("","t", "Number of time iterartions",false, 1,"int",cmd);
TCLAP::ValueArg<double> JArg("","Jc", "Ising interaction in the z-direction",false, 0.,"double",cmd);
TCLAP::ValueArg<double> JpArg("","Jp", "Ising interaction between A and B",false, 0.,"double",cmd);
TCLAP::ValueArg<double> DJs("","DJs", "Delta in the Js interacions on chain",false, 0.,"double",cmd);
TCLAP::ValueArg<double> Js("","Js", "Center of the Js interactions on chain",false, 0,"double",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> Dbs("","Dbs", "Delta in the magnetic field on spins",false, 0,"double",cmd);
TCLAP::ValueArg<int> one_state("","one_state", "State l",false, 0,"int",cmd);
TCLAP::ValueArg<int> ifrandom("","ifrandom", "0 if you dont want randstate",false,1,"int",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for k20, 1 for c20",false, 0,"int",cmd);
TCLAP::SwitchArg no_general_report("","no_general_report","Print the general report", cmd);
TCLAP::ValueArg<string> modelArg("","model", "Option" ,false,"nichts", "string",cmd);
TCLAP::ValueArg<int> tAvg("","t_ave", "Number of time averaged over",false, 1,"int",cmd);
TCLAP::ValueArg<int> xlenArg("","x", "Some number x",false, 0,"int",cmd);
TCLAP::ValueArg<int> symr("","symR", "If symmetries sectors of reflections are used",false, 0,"int",cmd);

int main(int argc,char* argv[]) {
  // Set initial stuff
  cout.precision(17);
  hipSetDevice(dev.getValue());
  itpp::RNG_randomize();
  cmd.parse(argc,argv);
  string option=optionArg.getValue();
  string model=modelArg.getValue();
  double J=JArg.getValue();
  double Jp=JpArg.getValue();
  int nqubits = nqubitsArg.getValue();
  int numt=numtArg.getValue();
  int xlen=xlenArg.getValue();
  
  
  int l=pow(2,nqubits);    
  int nqubits_env,xl;
  
  itpp::ivec conxA,conxB;
  
  //Se elige el modelo a usar
  void (*evolution)(double *, double *, itpp::vec, double, double, itpp::mat, int, int, itpp::ivec, itpp::ivec);
  if(model=="model1") {
    evolution=model::model1;
    nqubits_env=nqubits-1;
  }
  if(model=="model11") {
    evolution=model::model11;
    nqubits_env=nqubits-1;
  }  
  if(model=="model2") {
    evolution=model::model2;
    nqubits_env=nqubits-1;
  }
  if(model=="model3") {
    evolution=model::model3;
    nqubits_env=nqubits-1;
  }
  if(model=="model3_open") {
    evolution=model::model3_open;
    nqubits_env=nqubits-1;
  }
  if(model=="model3_open_op1") {
    evolution=extra_model::model3_open_op1;
    nqubits_env=nqubits-1;
  }
  if(model=="model3_open_op2") {
    evolution=extra_model::model3_open_op2;
    nqubits_env=nqubits-1;
  }
  if(model=="model3_open_op3") {
    evolution=extra_model::model3_open_op3;
    nqubits_env=nqubits-1;
  }  
  if(model=="model3_open_op4") {
    evolution=extra_model::model3_open_op4;
    nqubits_env=nqubits-1;
  }  
  if(model=="model3_open_op5") {
    evolution=extra_model::model3_open_op5;
    nqubits_env=nqubits-1;
  }  
  if(model=="model3_open_op6") {
    evolution=extra_model::model3_open_op6;
    nqubits_env=nqubits-1;
  }    
  if(model=="model4") {
    evolution=model::model4;
    nqubits_env=nqubits-1;
  }
  if(model=="model4_open") {
    evolution=model::model4_open;
    nqubits_env=nqubits-1;
  }
  if(model=="model5") {
    evolution=model::model5;
    nqubits_env=nqubits-1;
  }
  if(model=="model5_open") {
    evolution=model::model5_open;
    nqubits_env=nqubits-1;
  }
  if(model=="model7") {
    evolution=model::model7;
    nqubits_env=nqubits-1;
  }
  if(model=="model8") {
    evolution=model::model8;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar") {
    evolution=model::modelVar;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar1") {
    evolution=extra_model::modelVar1;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar2") {
    evolution=extra_model::modelVar2;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar3") {
    evolution=extra_model::modelVar3;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar4") {
    evolution=extra_model::modelVar4;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar5") {
    evolution=extra_model::modelVar5;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar6") {
    evolution=extra_model::modelVar6;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar7") {
    evolution=extra_model::modelVar7;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar8") {
    evolution=extra_model::modelVar8;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar9") {
    evolution=extra_model::modelVar9;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar10") {
    evolution=extra_model::modelVar10;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar11") {
    evolution=extra_model::modelVar11;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar12") {
    evolution=extra_model::modelVar12;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar13") {
    evolution=extra_model::modelVar13;
    nqubits_env=nqubits-1;
  }
  if(model=="modelVar14") {
    evolution=extra_model::modelVar14;
    nqubits_env=nqubits-1;
  }
  if(model=="modelConexComplete") {
    evolution=extra_model::modelConexComplete;
    nqubits_env=nqubits-1;
  }
  if(model=="modelConexRand") {
    evolution=extra_model::modelConexRand;
    nqubits_env=nqubits-1;
    
    ifstream con;
    con.open("conexiones.txt");
    int len, nada;
    
    con >> len >> nada;
    
    itpp::ivec conX(len);
    
    conxA=conX;
    conxB=conX;
    for(int i=0;i<len;i++) {
      con >> conxA(i) >> conxB(i);
    }
    con.close();
  }
  if(model=="modelConexRandB") {
    evolution=extra_model::modelConexRandB;
    nqubits_env=nqubits-1;
    
    ifstream con;
    con.open("conexiones.txt");
    int len, nada;
    
    con >> len >> nada;
    
    itpp::ivec conX(len);
    
    conxA=conX;
    conxB=conX;
    for(int i=0;i<len;i++) {
      con >> conxA(i) >> conxB(i);
    }
    con.close();
  }  
  if(model=="modelConexRandABC") {
    evolution=extra_model::modelConexRandABC;
    nqubits_env=nqubits-1;
    
    ifstream con;
    con.open("conexiones.txt");
    int len, nada;
    
    con >> len >> nada;
    
    itpp::ivec conX(len);
    
    conxA=conX;
    conxB=conX;
    for(int i=0;i<len;i++) {
      con >> conxA(i) >> conxB(i);
    }
    con.close();
  }
  if(model=="model3_open_VarMagnetic") {
    evolution=extra_model::model3_open_VarMagnetic;
    nqubits_env=nqubits-1;
  }

  
  int Cseed=CseedArg.getValue();int PARAMseed=PARAMseedArg.getValue();int Eseed=EseedArg.getValue();
  
  if (Cseed == 0 ){
    Random seed_uran1; 
    Cseed=seed_uran1.strong();
  }
  itpp::RNG_reset(Cseed);
  //ESTADO INICIAL C
  itpp::cvec cstate = itppextmath::RandomState(2);
  
  if (Cseed == -1 ){
    cstate = itpp::ones_c(2);
    cstate=cstate*(1/sqrt(2));
  }
  
  if (Eseed == 0 ){
    Random seed_uran3; 
    Eseed=seed_uran3.strong();
  }
  itpp::RNG_reset(Eseed);
  
  itpp::cvec state;
  if(xlen==0) {
    itpp::cvec estate = itppextmath::RandomState(l/2);
  
    //Preparacion estado inicial
    state=tensor_prod(cstate,estate);
  }
  else {
    xl=pow(2,xlen);
    itpp::cvec estateA = itppextmath::RandomState(xl);
    itpp::cvec estateB = itppextmath::RandomState(l/(xl*2));
  
    //Preparacion estado inicial
    state=tensor_prod(cstate,tensor_prod(estateB,estateA)); 
  }
  
  //PARAMETROS SEED
  if (PARAMseed == 0 ){
    Random seed_uran2; 
    PARAMseed=seed_uran2.strong();
  }
  itpp::RNG_reset(PARAMseed);
  itpp::vec js = itpp::ones(nqubits_env)*(Js.getValue()-DJs.getValue()) + itpp::randu(nqubits_env)*(2*DJs.getValue());
  //cout<<js<<endl;
  
  itpp::vec b_one(3); b_one(0)=bx.getValue(); b_one(1)=by.getValue(); b_one(2)=bz.getValue();
  //CAMPO MAGNETICO NO UNIFORME
  itpp::mat b(nqubits,3);
  if (Dbs.getValue() == -1) {
    //Caso de campo paralelo
    for(int i=0;i<nqubits;i++) { 
      b(i,0)=b_one(0);
      b(i,1)=0;
      b(i,2)=b_one(2);
      if (i<xlen) {
	b(i,0)=0;
      }
    }
  }
  else if (Dbs.getValue() == -2) {
    //Caso de campo perpendicular
    for(int i=0;i<nqubits;i++) { 
      b(i,0)=b_one(0);
      b(i,1)=0;
      b(i,2)=b_one(2);
      if (i<xlen) {
	b(i,2)=0;
      }
    }
  }  
  else {
    for(int i=0;i<nqubits;i++) { 
      b(i,0)=b_one(0)-Dbs.getValue() + itpp::randu()*2*Dbs.getValue();
      b(i,1)=0;
      b(i,2)=b_one(2)-Dbs.getValue() + itpp::randu()*2*Dbs.getValue();
    }
  }
    
  //itpp::cvec state = itppextmath::RandomState(l);
  
  //Comprobando que sea unitario al principio
  //cout<<"NORMA "<<itpp::norm(state)<<endl;
  
  
  //Se sube el estado al dev
  double *dev_R,*dev_I;
  evcuda::itpp2cuda_malloc(state,&dev_R,&dev_I);
  
  if(option=="purity") {
    for(int it=0;it<numt;it++) {
      cout<<std::real(evmath::purity_last_qubit(state,l))<<" ";
      
      //       itpp::cmat rho = evmath::reduced_densMat(dev_R,dev_I,l/2-1,nqubits);
      //       rho=rho*rho;
      //       
      //       cout<<itpp::trace(rho)<<endl;
      
      evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
      
      evcuda::cuda2itpp(state,dev_R,dev_I);
      
      
    }
    cout<<endl;
  }
  
  if(option=="purity_onet") {
    for(int it=0;it<numt;it++) {
      evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
    }
    evcuda::cuda2itpp(state,dev_R,dev_I);
    cout<<std::real(evmath::purity_last_qubit(state,l))<<endl;
  }
  
    if(option=="densMat") {
    for(int it=0;it<numt;it++) {
      evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
    }
    evcuda::cuda2itpp(state,dev_R,dev_I);
    itpp::cmat rho= itpp::zeros_c(2,2); 
    itpp::cvec a=state.right(l/2);
    itpp::cvec b=state.left(l/2); 
  
    rho(0,0)=itpp::dot(a,itpp::conj(a));
    rho(0,1)=itpp::dot(a,itpp::conj(b));
    rho(1,0)=itpp::dot(b,itpp::conj(a));
    rho(1,1)=itpp::dot(b,itpp::conj(b));
    cout<<rho(0,0)<<" "<<rho(0,1)<<" "<<rho(1,0)<<" "<<rho(1,1)<<endl;
    
    rho=rho*rho;
//     cout<<"P = "<<itpp::trace(rho);
  }
  
  if(option=="purity_gamma") {
    itpp::cvec zerostate=state;
    int div=150;
    for(int gi=0;gi<=div;gi++) {
      double Jpi=((itpp::pi*gi)/(div+1))/sqrt(Jp);
      evcuda::itpp2cuda(zerostate,dev_R,dev_I);
      for(int it=0;it<numt;it++) {
        evolution(dev_R,dev_I,js,J,Jpi,b,nqubits,xlen,conxA,conxB);
      }
      cudaCheckError("valio verga",383);
      evcuda::cuda2itpp(state,dev_R,dev_I);
      cout<<std::real(evmath::purity_last_qubit(state,l))<<endl;
    }
  }
  
  if(option=="purity_all_systems") {
    int whichA,whichB,whichC;
    itpp::cmat rhoA,rhoB;
    for(int it=0;it<numt;it++) {
      //cout<<std::real(evmath::purity_last_qubit(state,l))<<endl;
      
      whichC=l/2;
      whichA=xl-1;
      whichB=(l-1)^(whichA^whichC);
            
      rhoA = evmath::reduced_densMat(dev_R,dev_I,whichA,nqubits);
      rhoB = evmath::reduced_densMat(dev_R,dev_I,whichB,nqubits);
      //rhoC = evmath::reduced_densMat(dev_R,dev_I,whichC,nqubits);
      
      rhoA=rhoA*rhoA;
      rhoB=rhoB*rhoB;
      //rhoC=rhoC*rhoC;

      cout<<real(trace(rhoA))<<" "<<real(itpp::trace(rhoB))<<" "<<real(evmath::purity_last_qubit(state,l))<<endl;
      
      evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
      
      evcuda::cuda2itpp(state,dev_R,dev_I);
      
      
    }
  }
  
  if(option=="purity_timeavg") {
    itpp::vec purities(100);
    itpp::cvec zerostate=state;
    int div=300;
    for(int ij=0;ij<=div;ij++) {
      double Ji=((itpp::pi*ij)/div);
      evcuda::itpp2cuda(zerostate,dev_R,dev_I);
      for(int i=0;i<100;i++) {
	evolution(dev_R,dev_I,js,J,Ji,b,nqubits,xlen,conxA,conxB);
      }
      for(int i=0;i<100;i++) {
	evcuda::cuda2itpp(state,dev_R,dev_I);
	evolution(dev_R,dev_I,js,J,Ji,b,nqubits,xlen,conxA,conxB);
	purities(i)=std::real(evmath::purity_last_qubit(state,l));
      }
      cout<<Ji<<" "<<itpp::mean(purities)<<" "<<std::sqrt(itpp::variance(purities))<<endl;    
    }
    cout<<endl;
  }
  
  if(option=="test_Umat") {
    itpp::cmat U;
    if(symr.getValue()==1) {
      U = evmath::evolution_matrix_chain_reflection(J,b.get_row(0),nqubits,1);
    }
    else { 
      U = evmath::evolution_matrix_chain_translation(J,b.get_row(0),nqubits,1);
    }
    int rcont = U.rows();
    //Prueba unitariedad
    cout<<itpp::norm(itpp::eye_c(rcont)-U*itpp::hermitian_transpose(U))<<endl;
    //Prueba de evoluciones
    itpp::cvec state2 = U * state;
    evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
    evcuda::cuda2itpp(state,dev_R,dev_I);
    cout<<itpp::norm(state-state2)<<endl;  
  }
  
  if(option=="get_spectra") {
    itpp::cmat U;
    if(symr.getValue()==0) {
      U = evmath::evolution_matrix_chain_reflection(J,b.get_row(0),nqubits,1);
    }
    else { 
      U = evmath::evolution_matrix_chain_translation(J,b.get_row(0),nqubits,1);
    }
    int rcont = U.rows();
    itpp::cvec eigenvalues(rcont);
    itpp::cmat eigenvectors(rcont,rcont);
    itpp::eig(U,eigenvalues,eigenvectors);
    
    for(int i=0;i<rcont;i++) {
      cout<<argument(eigenvalues(i))<<endl;
    }
    
    //Calculo del error
    //double error=itpp::norm(U-eigenvectors*itpp::diag(eigenvalues)*itpp::hermitian_transpose(eigenvectors));
    //cout<<"ERROR "<<error<<endl;
  }  
  
  if(option=="correlation") {
    double *dev_sumdxR,*dev_sumdxI;
    evcuda::cmalloc(&dev_sumdxR,&dev_sumdxI,l);
    double *dev_inR,*dev_inI;
    evcuda::cmalloc(&dev_inR,&dev_inI,l);
    
    evcuda::itpp2cuda(state,dev_R,dev_I);
    
    int numthreads;
    int numblocks;
    choosenumblocks(l,numthreads,numblocks); 
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits,l);
    itpp::cvec sumstate(l);
    
    evcuda::cuda2itpp(sumstate,dev_sumdxR,dev_sumdxI);
    evcuda::cuda2itpp(state,dev_inR,dev_inI);
    
    
    for(int it=0;it<numt;it++) {
      sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits,l);
      evcuda::cuda2itpp(sumstate,dev_sumdxR,dev_sumdxI);
      evcuda::cuda2itpp(state,dev_inR,dev_inI);
      
      evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
      evolution(dev_inR,dev_inI,js,J,Jp,b,nqubits,xlen,conxA,conxB);
      
      cout<<sqrt(std::norm(itpp::dot(itpp::conj(sumstate),state)))/nqubits<<endl;
    }
    
    hipFree(dev_sumdxR);
    hipFree(dev_sumdxI);
    
  }
  
  if(option=="corr_map") {
    double *dev_sumdxR,*dev_sumdxI;
    evcuda::cmalloc(&dev_sumdxR,&dev_sumdxI,l);
    double *dev_inR,*dev_inI;
    evcuda::cmalloc(&dev_inR,&dev_inI,l);
    
    itpp::vec correlations(20);
    itpp::cvec zerostate=state;
    itpp::cvec sumstate=state;
    evcuda::itpp2cuda(state,dev_R,dev_I);
    
    int numthreads;
    int numblocks;
    choosenumblocks(l,numthreads,numblocks); 
    
    int div=250;
    double bxi,bzi;
    for(int idiv=0;idiv<div;idiv++) {
      bxi=((2*itpp::pi*idiv)/div);
      for(int jdiv=0;jdiv<div;jdiv++) {
	evcuda::itpp2cuda(zerostate,dev_R,dev_I);
	sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits,l);
	
	bzi=((2*itpp::pi*jdiv)/div);
	b(0)=bxi; b(2)=bzi;
	
	for(int i=0;i<80;i++) {
	  evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
	  evolution(dev_inR,dev_inI,js,J,Jp,b,nqubits,xlen,conxA,conxB);
	}
	
	for(int i=0;i<20;i++) {
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits,l);
	  
	  evcuda::cuda2itpp(sumstate,dev_sumdxR,dev_sumdxI);
	  evcuda::cuda2itpp(state,dev_inR,dev_inI);
	  
	  evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
	  evolution(dev_inR,dev_inI,js,J,Jp,b,nqubits,xlen,conxA,conxB);
	  
	  correlations(i)=sqrt(std::norm(itpp::dot(itpp::conj(sumstate),state)))/nqubits;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;    
      }
    }
    
    hipFree(dev_sumdxR);
    hipFree(dev_sumdxI);
  }
  
  if(option=="trU") {
    
    itpp::cvec stateBra=state; 
    for(int t=0;t<numt;t++) {
      evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
      evcuda::cuda2itpp(state,dev_R,dev_I);
      cout<<t+1<<" "<<norm(itpp::dot(itpp::conj(stateBra),state))<<endl;  
    }
  }
  
  
  
  if(option=="carlostest_chain") {
    itpp::cvec cstate = state;
    //CUDA evolution
    evolution(dev_R,dev_I,js,J,Jp,b,nqubits,xlen,conxA,conxB);
    evcuda::cuda2itpp(state,dev_R,dev_I);
    //Carlos evolution
    for(int i=0;i<nqubits;i++) {
      spinchain::apply_ising_z(cstate,js(i),i,(i+1)%nqubits);
    }
    for(int i=0;i<nqubits;i++) {
      spinchain::apply_magnetic_kick(cstate,b_one,i);
    }
    
    cout<<itpp::norm(state-cstate)<<endl; 
  }
  
  if(option=="carlostest_sumdx") {
    double *dev_sumdxR,*dev_sumdxI;
    evcuda::cmalloc(&dev_sumdxR,&dev_sumdxI,l);
    
    int numthreads;
    int numblocks;
    choosenumblocks(l,numthreads,numblocks);
    
    itpp::cvec cstate = state;
    
    //CUDA evolution
    //evcuda::apply_sumdx(nqubits,dev_R,dev_I,dev_sumdxR,dev_sumdxI);
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits,l);
    evcuda::cuda2itpp(state,dev_sumdxR,dev_sumdxI);
    //Carlos evolution
    itpp::cmat sumsig=itpp::zeros_c(l,l);
    b_one(0)=1.; b_one(1)=0.; b_one(2)=0.;
    
    for(int i=0;i<nqubits;i++) {
      sumsig+=itppextmath::sigma(b_one,i,nqubits);
    }
    cstate=sumsig*cstate;
    cout<<itpp::norm(state-cstate)<<endl; 
    cout<<itpp::norm(state)<<endl;
    cout<<itpp::norm(cstate)<<endl;
    
    
    hipFree(dev_sumdxR);
    hipFree(dev_sumdxI);
    
  }
  
  hipFree(dev_R);
  hipFree(dev_I);
  
  
}




