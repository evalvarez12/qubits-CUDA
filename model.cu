#include "hip/hip_runtime.h"
#ifndef MODEL
# define MODEL
# include "tools.cpp"
# include "cuda_functions.cu"
namespace model{
 
void model1(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*      MODEL 1
             *
           *   *   
          *     * -- * last qubit
           *   *
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits-1),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion 0 con ultimo
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model11(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*      MODEL 1.1
             *
           * | *   
          *  |  * -- * last qubit
           * | *
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la interaccion cruzada dentro de la cadena
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-2,2,dev_R,dev_I,cos(js(nqubits/2)),sin(js(nqubits/2)),l);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits-1),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion 0 con ultimo
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  

void model2(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*      MODEL 2
             *
           *   * - 
          *     * - * last qubit
           *   * - 
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits-1),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion 0,1,penultimo con ultimo
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,1,dev_R,dev_I,cos(j),sin(j),l);
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,nqubits-2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
  
void chain(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*    MODEL CHAIN CLOSED
             *
           *   * 
          *     * 
           *   * 
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
  
  
void chain_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*    MODEL CHAIN OPEN

    *  *  *  *  *  *  *  *  *  
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void lattice(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL LATTICE
       
       *   *   *   *
       *   *   *   *
       *   *   *   *
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  int i_hor,i_ver;
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    i_hor=(i+1)%xlen+(i/xlen)*xlen;
    i_ver=(i+xlen)%nqubits;
    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model3(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 3 CLOSED
       
   <- *   *   *   *   *   *   *  ->
             /      
     <- *   *   *   * -> 
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(xlen),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-1-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,(i+1)%(nqubits-1-xlen)+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model3_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model4(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 4 CLOSED
       
   <- *   *   *   *   *   *   *  ->
       \     /   /   /
     <- *   *   *   * -> 
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(xlen),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-1-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,(i+1)%(nqubits-1-xlen)+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //las interacciones A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-4,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-2,xlen+2,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen+4,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model4_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 4 OPEN
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen/2-1,nqubits-xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen/2+1,nqubits-xlen+1,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,nqubits-2,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,xlen/2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model5(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 5 CLOSED
       
  <- *   *   *   *   *   *  ->
      \   \   \   \   \   \
   <-  *   *   *   *   *   * ->
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(xlen),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-1-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,(i+1)%(nqubits-1-xlen)+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B es 1 a 1 
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+xlen,dev_R,dev_I,cos(jp),sin(jp),l);
    } 
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
  
void model5_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 5 OPEN
       
  *   *   *   *   *   * 
   \   \   \   \   \   \
    *   *   *   *   *   *
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B es 1 a 1 
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+xlen,dev_R,dev_I,cos(jp),sin(jp),l);
    } 
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model6(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 6 CLOSED
 <-*   *   *   *   *   *  ->    
    \     /     \     /
  <- *   *   *   *   *   *  ->
      \       \     /     \
   <-  *   *   *   *   *   * ->
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(xlen),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-1-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,(i+1)%(nqubits-1-xlen)+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B es 1 a 1 
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+xlen,dev_R,dev_I,cos(jp),sin(jp),l);
    } 
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
  
void model7(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 7 CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,9,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,15,dev_R,dev_I,cos(jp),sin(jp),l);
  //CONEXIONES EXTRA A B
  //Ui_kernel<<<numblocks,numthreads>>>(0,7,dev_R,dev_I,cos(jp),sin(jp),l);
  //Ui_kernel<<<numblocks,numthreads>>>(1,8,dev_R,dev_I,cos(jp),sin(jp),l);
  //Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  //Ui_kernel<<<numblocks,numthreads>>>(3,11,dev_R,dev_I,cos(jp),sin(jp),l);
  //Ui_kernel<<<numblocks,numthreads>>>(4,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,14,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
  
void model8(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 8 CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=10 B=15       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,12,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,15,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,17,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,18,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(6,19,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(7,21,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(8,23,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(9,25,dev_R,dev_I,cos(jp),sin(jp),l);
  //INTERACCIONES EXTRA A B
/*  Ui_kernel<<<numblocks,numthreads>>>(0,11,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,13,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,16,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,16,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,19,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(6,20,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(7,20,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(8,22,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(9,24,dev_R,dev_I,cos(jp),sin(jp),l); */ 
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,6,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void modelVar(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL VARIABLE CASO ESPECIAL
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit - not kicked
         PARA A=6 B=10       
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion variable  A B  
  Ui_kernel<<<numblocks,numthreads>>>(0,6,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(1,7,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(2,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(3,10,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(4,14,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(5,15,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits-1;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
    
 
} 

#endif    