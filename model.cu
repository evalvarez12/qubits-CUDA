#include "hip/hip_runtime.h"
#ifndef MODEL
# define MODEL
# include "tools.cpp"
# include "cuda_functions.cu"
namespace model{
 
void model1(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*      MODEL 1
             *
           *   *   
          *     * -- * last qubit
           *   *
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits-1),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion 0 con ultimo
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model11(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*      MODEL 1.1
             *
           * | *   
          *  |  * -- * last qubit
           * | *
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la interaccion cruzada dentro de la cadena
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-2,2,dev_R,dev_I,cos(js(nqubits/2)),sin(js(nqubits/2)),l);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits-1),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion 0 con ultimo
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  

void model2(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*      MODEL 2
             *
           *   * - 
          *     * - * last qubit
           *   * - 
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits-1),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //se hace la interacion 0,1,penultimo con ultimo
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,1,dev_R,dev_I,cos(j),sin(j),l);
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,nqubits-2,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
  
void chain(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*    MODEL CHAIN CLOSED
             *
           *   * 
          *     * 
           *   * 
             *
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(nqubits),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
  
  
void chain_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int extra){ 
  /*    MODEL CHAIN OPEN

    *  *  *  *  *  *  *  *  *  
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void lattice(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL LATTICE
       
       *   *   *   *
       *   *   *   *
       *   *   *   *
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  int i_hor,i_ver;
  choosenumblocks(l,numthreads,numblocks);
  for(int i=0;i<nqubits-1;i++) {
    i_hor=(i+1)%xlen+(i/xlen)*xlen;
    i_ver=(i+xlen)%nqubits;
    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }
  
void model3(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 3 CLOSED
       
   <- *   *   *   *   *   *   *  ->
             /      
     <- *   *   *   * -> 
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(xlen),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-1-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,(i+1)%(nqubits-1-xlen)+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model3_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 3 OPEN
       
      *   *   *   *   *   *   *  
             /      
        *   *   *   * 
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model4(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 4 CLOSED
       
   <- *   *   *   *   *   *   *  ->
       \     /   /   /
     <- *   *   *   * -> 
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%(xlen),dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-1-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,(i+1)%(nqubits-1-xlen)+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //las interacciones A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-4,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-2,xlen+2,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen+4,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  } 
  
void model4_open(double *dev_R, double *dev_I, itpp::vec js, double j, double jp, itpp::mat b , int nqubits, int xlen){ 
  /*    MODEL 4 OPEN
       
  *   *   *   *   *   *   *
       \     /   /   /
        *   *   *   * 
             \      
              *  last qubit
          
  */
  int numthreads, numblocks;
  double kcos,ksin,bx,by,bz;
  int l=pow(2,nqubits);
  choosenumblocks(l,numthreads,numblocks);
  //la evolucion de la cadena A de tamaño xlen
  for(int i=0;i<xlen-1;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i,i+1,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }  
  //la evolucion de la cadena B de tamaño nqubits - xlen - 1  
  for(int i=0;i<nqubits-2-xlen;i++) {
    Ui_kernel<<<numblocks,numthreads>>>(i+xlen,i+1+xlen,dev_R,dev_I,cos(js(i)),sin(js(i)),l);
    }
  //la interaccion A B  
  Ui_kernel<<<numblocks,numthreads>>>(xlen-4,xlen,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-2,xlen+2,dev_R,dev_I,cos(jp),sin(jp),l);
  Ui_kernel<<<numblocks,numthreads>>>(xlen-1,xlen+4,dev_R,dev_I,cos(jp),sin(jp),l);
  //se hace la interacion 0 con A
  Ui_kernel<<<numblocks,numthreads>>>(nqubits-1,0,dev_R,dev_I,cos(j),sin(j),l);
  //evolucion patada magnetica
  for(int i=0;i<nqubits;i++) {
    set_parameters(b.get_row(i),kcos,ksin,bx,by,bz);
    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx,by,bz,kcos,ksin,l);     
    }
  return;  
  }  
    
    
} 

#endif    