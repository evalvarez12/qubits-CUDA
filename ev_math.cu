#include "hip/hip_runtime.h"
#ifndef EVMATH
# define EVMATH
namespace evmath{

typedef void (*operador_evolucion)(double *, double *, itpp::vec, double, double, itpp::mat, int, int, itpp::ivec, itpp::ivec);

itpp::cmat evolution_matrix(operador_evolucion evolucion, itpp::vec js, double j, double jp, itpp::mat b ,  int nqubits, int extra, itpp::ivec A, itpp::ivec B)  {
  int l = pow(2,nqubits);
  double *dev_umatR,*dev_umatI;
  evcuda::cmalloc(&dev_umatR,&dev_umatI,l);
  int numthreads, numblocks;
  choosenumblocks(l,numthreads,numblocks);
  itpp::cvec state(l);
  itpp::cmat Umat(l,l);
  for(int i=0;i<l;i++) {
    index_one<<<numblocks,numthreads>>>(dev_umatR,dev_umatI,l,i);
    evolucion(dev_umatR,dev_umatI,js,j,jp,b,nqubits,extra,A,B);
    evcuda::cuda2itpp(state,dev_umatR,dev_umatI);
    Umat.set_col(i,state);
  }
  hipFree(dev_umatR);
  hipFree(dev_umatI);
  return Umat;
}

itpp::cmat evolution_matrix(operador_evolucion evolucion, itpp::vec js, double j, double jp, itpp::mat b ,  int nqubits, int extra, itpp::ivec A, itpp::ivec B, int symr)  { 
  //SECTORES DE SIMETRIA POR REFLEXION
  int l = pow(2,nqubits);
  double *dev_umatR,*dev_umatI;
  evcuda::cmalloc(&dev_umatR,&dev_umatI,l);
  
  int *S=new int[l];
  for(int m=0;m<l;m++) {
    S[m]=2;
  }
  find_states_reflection(S,nqubits,symr,l);
  int rcont=0;
  
  for(int i=0;i<l;i++) {
    if(S[i]==1) {
      S[rcont]=i;
      rcont++;
    }
  }
  
  int *dev_S;
  hipMalloc((void**)&dev_S,rcont*sizeof(int)); 
  hipMemcpy(dev_S,S,rcont*sizeof(int),hipMemcpyHostToDevice);
  double *dev_dotR,*dev_dotI;
  evcuda::cmalloc(&dev_dotR,&dev_dotI,rcont);
  int numthreads, numblocks;
  choosenumblocks(l,numthreads,numblocks);
  
  itpp::cvec state(rcont);
  itpp::cmat Umat(rcont,rcont);
  for(int i=0;i<rcont;i++) {
    to_zero<<<numblocks,numthreads>>>(dev_umatR,dev_umatI,l); 
    reflection_proyector<<<1,1>>>(dev_umatR,dev_umatI,nqubits,symr,S[i]);
    times_norm<<<numblocks,numthreads>>>(dev_umatR,dev_umatI,l); 
  
    evolucion(dev_umatR,dev_umatI,js,j,jp,b,nqubits,extra,A,B);
   
    to_zero<<<numblocks,numthreads>>>(dev_dotR,dev_dotI,rcont); 
    proyected_dot_reflection<<<numblocks,numthreads>>>(dev_umatR,dev_umatI,dev_dotR,dev_dotI,nqubits,rcont,symr,dev_S);
    evcuda::cuda2itpp(state,dev_dotR,dev_dotI);
    
    Umat.set_col(i,state);
  }
  hipFree(dev_umatR);
  hipFree(dev_umatI);
  return Umat;
}


std::complex<double> purity_last_qubit(itpp::cvec state, int l) {
  itpp::cmat rho= itpp::zeros_c(2,2); 
  itpp::cvec a=state.right(l/2);
  itpp::cvec b=state.left(l/2); 
  
  rho(0,0)=itpp::dot(a,itpp::conj(a));
  rho(0,1)=itpp::dot(a,itpp::conj(b));
  rho(1,0)=itpp::dot(b,itpp::conj(a));
  rho(1,1)=itpp::dot(b,itpp::conj(b));
  rho=rho*rho;
  return itpp::trace(rho);
}

itpp::cmat reduced_densMat (double* dev_R, double* dev_I, int which, int nqubits) { 
  int ndens=pow(2,numbits(which));
  int l=pow(2,nqubits);
  double *densR=new double[ndens*ndens];
  double *densI=new double[ndens*ndens];
  double *dev_densR;      
  double *dev_densI;
  evcuda::cmalloc(&dev_densR,&dev_densI,ndens*ndens);
  cudaCheckError("dot",0);
  int numthreads, numblocks;
  choosenumblocks(l,numthreads,numblocks);
  to_zero<<<numthreads,numblocks>>>(dev_densR,dev_densI,ndens*ndens); 
  itpp::cmat densMat(ndens,ndens);
  //cout<<numblocks<<"  "<<numthreads/ndens<<endl;
  for(int i=0;i<ndens;i++) {
    for(int j=0;j<ndens;j++) {
      density_matrix<<<numblocks,numthreads>>>(which,ndens,i,j,dev_R,dev_I,dev_densR,dev_densI,l/ndens);
      cudaCheckError("dot",i+j);
    }
  }
  hipMemcpy(densR,dev_densR,ndens*ndens*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(densI,dev_densI,ndens*ndens*sizeof(double),hipMemcpyDeviceToHost);
  for(int i=0;i<ndens;i++) {
    for(int j=0;j<ndens;j++) {
      densMat(i,j)=std::complex<double>(densR[(ndens*i)+j],densI[(ndens*i)+j]);
    }
  }  
  hipFree(dev_densR);
  hipFree(dev_densI);
  return densMat;
}

void apply_sumdx(int nqubits,double* dev_R,double* dev_I,double* dev_sumdxR,double* dev_sumdxI) {
  int l=pow(nqubits,2);
  int numthreads;
  int numblocks;
  choosenumblocks(l,numthreads,numblocks); 
  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits,l);
  cudaCheckError("kick",10);
}

} 

#endif    