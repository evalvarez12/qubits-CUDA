#include "hip/hip_runtime.h"
// Includes {{{
#include <iostream>
#include <itpp/itbase.h>
#include <spinchain.cpp>
#include <dev_random.cpp>
#include <itpp_ext_math.cpp>
#include <math.h>
#include <tclap/CmdLine.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "tools.cpp"
#include "cuda_functions.cu"
#include "cuda_utils.cu"
#include "ev_routines.cu"
#include "cfp_routines.cu"
#include <time.h>
// }}}
// TCLAP setup {{{
TCLAP::CmdLine cmd("Command description message", ' ', "0.1");
TCLAP::ValueArg<unsigned int> seed("s","seed", "Random seed [0 for urandom]",false, 243243,"unsigned int",cmd);
TCLAP::ValueArg<string> optionArg("o","option", "Option" ,false,"nichts", "string",cmd);
TCLAP::ValueArg<int> nqubits("q","qubits", "Number of qubits",false, 3,"int",cmd);
TCLAP::ValueArg<int> numt("","t", "Number of time iterartions",false, 1,"int",cmd);
TCLAP::ValueArg<int> position("","position", "The position of something",false, 0,"int",cmd);
TCLAP::ValueArg<int> whichq("","which", "Which qubits in densmat",false, 1,"int",cmd);
TCLAP::ValueArg<int> x("","x", "Size of the x-dimention",false, 0,"int",cmd);
// TCLAP::ValueArg<int> y("","y", "Size of the y-dimention",false, 0,"int",cmd);
//TCLAP::ValueArg<int> position2("","position2", "The position of something",false, 3,"int",cmd);
TCLAP::ValueArg<double> ising("","ising_z", "Ising interaction in the z-direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> deltav("","delta", "Some small delta",false, 1,"double",cmd);
TCLAP::ValueArg<int> trotternum("","trotter", "Number of steps for trotter-suzuki algorithm",false, 1,"int",cmd);
TCLAP::ValueArg<double> bx("","bx", "Magnetic field in x direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> by("","by", "Magnetic field in y direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> bz("","bz", "Magnetic field in z direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> beginx("","startx", "Magnetic field start in x direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> beginz("","startz", "Magnetic field start in z direction",false, 0,"double",cmd);
TCLAP::ValueArg<double> km("","k", "Momentum of the proyector",false,0,"double",cmd);
TCLAP::ValueArg<int> one_state("","one_state", "State l",false, 0,"int",cmd);
TCLAP::ValueArg<int> ifrandom("","ifrandom", "0 if you dont want randstate",false,1,"int",cmd);
TCLAP::ValueArg<int> dev("","dev", "Gpu to be used, 0 for k20, 1 for c20",false, 0,"int",cmd);
  TCLAP::SwitchArg no_general_report("","no_general_report",
      "Print the general report", cmd);
// }}}
double diffclock(clock_t clock1,clock_t clock2) // {{{
{
  double diffticks=clock1-clock2;
  double diffms=(diffticks*1000)/CLOCKS_PER_SEC;
  return diffms;
}  // }}}


// }}}
int main(int argc,char* argv[]) {
  // Setup CUDA devide, random numbers, command line parserc, and other parametrs {{{
  hipSetDevice(dev.getValue());
//   itpp::RNG_randomize();
double error=0;
  cmd.parse(argc,argv);
  // {{{ Set seed for random
  unsigned int semilla=seed.getValue();
  
//   std::cout << "La semilla es " << semilla << endl;

  if (semilla == 0){
    Random semilla_uran; semilla=semilla_uran.strong();
  } 
  itpp::RNG_reset(semilla);
  // }}}
  // {{{ Report on the screen
  if(!no_general_report.getValue()){
    cout << "#linea de comando: "; 
    for(int i=0;i<argc;i++){ 
      cout <<argv[i]<<" " ;
    } cout << endl ;
    cout << "#semilla = " << semilla << endl; 
    error += system("echo \\#hostname: $(hostname)");
    error += system("echo \\#comenzando en: $(date)");
    error += system("echo \\#uname -a: $(uname -a)");
    error += system("echo \\#working dir: $(pwd)");
  }
  // }}}
  string option=optionArg.getValue();
  
  int l=pow(2,nqubits.getValue());
  int numthreads, numblocks;
  choosenumblocks(l,numthreads,numblocks);
  int div=choosediv(nqubits.getValue());
  // }}}
  // Create workspace in the CPU
  double *R=new double[l], *I=new double[l];
  // Create random state {{{
  
  double *dev_R, *dev_I;
  randomstate(l,R,I);

  if (ifrandom.getValue()!=1) {
    for(int i=0;i<l;i++) {
      R[i]=0;
      I[i]=0;
    }
    R[one_state.getValue()]=1;
  }

  
  cudaSafeCall(hipMalloc((void**)&dev_R,l*sizeof(double)),"malloc",124);
  cudaSafeCall(hipMalloc((void**)&dev_I,l*sizeof(double)),"malloc",125);
  
  cudaSafeCall(hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice),"memcpy",127);
  cudaSafeCall(hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice),"memcpy",128);
  // }}}
  if (option=="test_apply_ising") { // {{{
    double mcos=cos(ising.getValue());
    double msin=sin(ising.getValue());
    for(int n=0;n<numt.getValue();n++) {
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,mcos,msin,l);
      }
    }
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    hipFree(dev_R);
    hipFree(dev_I);
    for(int i=0;i<l;i++) {
      std::cout<<R[i]<<"  i"<<I[i]<<endl;
    }
  } // }}}
  if (option=="test_apply_magnetic_kick") { // {{{
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double mcos=cos(theta);
    double msin=sin(theta);
    for(int n=0;n<numt.getValue();n++) {
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,mcos,msin,l);
      }
    }
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      std::cout<<R[i]<<"  i"<<I[i]<<endl;
    }
  } // }}}
  if (option=="apply_chain") {  // {{{
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);    
    for(int n=0;n<numt.getValue();n++) {
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	cudaCheckError("ising",i);
      }
      
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	cudaCheckError("kick",i);
      }
    }
    
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      std::cout<<R[i]<<"  i"<<I[i]<<endl;
    }
  } // }}}
  if (option=="measure_time") { // {{{
    hipEvent_t kstart, kstop;
//     hipEvent_t cstart, cstop, kstart, kstop;
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);    
    float timek=0;
    hipEventCreate(&kstart);
    hipEventCreate(&kstop);
    hipEventRecord(kstart,0);
    clock_t begin=clock();
    //system("echo \\#Comenzando a hacer varias iteraciones:    $(date)");
    //cout << "Iteraciones son " << numt.getValue() << endl;
    for(int n=0;n<numt.getValue();n++) {
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	cudaCheckError("ising",i);
      }
      
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	cudaCheckError("kick",i);
      }
    }
    
    //system("echo \\#Terminando a hacer varias iteraciones:    $(date)");
    hipEventRecord(kstop,0);
    hipEventSynchronize(kstop);
    hipEventElapsedTime(&timek,kstart,kstop);
    clock_t end=clock();
    double tiempo=double(diffclock(end,begin));
    std::cout<<"CUDA EVENT "<<timek/(numt.getValue()*100)<<endl;
    cout <<"C clock "<<tiempo/(numt.getValue()*100)<< endl;
  } // }}}
  if (option=="check_inverse") { // {{{
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    itpp::cvec initialstate(l);
    itpp::cvec finalstate(l);
    for(int i=0;i<l;i++) {
      initialstate(i)=std::complex<double>(R[i],I[i]);
    }
    //se aplica la U
    for(int t=0;t<numt.getValue();t++) {
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	cudaCheckError("ising",i);
      }
      
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	cudaCheckError("kick",i);
      }
    }
    //Se aplica U^-1
    for(int t=0;t<numt.getValue();t++) {
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	cudaCheckError("kick",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,-1*isin,l);
	cudaCheckError("ising",i);
      }
    }
    
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      finalstate(i)=std::complex<double>(R[i],I[i]);
    }
    cout<<std::norm(itpp::dot(itpp::conj(initialstate),finalstate))<<endl;
  } // }}}
  if (option=="check_inverse_trotter2d") { // {{{
    int xlen=x.getValue();
    int num_trotter=trotternum.getValue();
    double delta=1./num_trotter;
    cout << delta << endl;
    int i_hor,i_ver;
    double icos=cos((delta/2.)*ising.getValue());
    double isin=sin((delta/2.)*ising.getValue());
    double theta=(delta)*sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=delta*bx.getValue()/theta;
    double by2=delta*by.getValue()/theta;
    double bz2=delta*bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    itpp::cvec initialstate(l);
    itpp::cvec finalstate(l);
    for(int i=0;i<l;i++) {
      initialstate(i)=std::complex<double>(R[i],I[i]);
    }
    //se aplica la U
    for(int t=0;t<numt.getValue();t++) {
      for(int it=0;it<num_trotter;it++) {
	for(int i=0;i<nqubits.getValue();i+=2) {
	  i_hor=(i+1)%xlen+(i/xlen)*xlen;
	  i_ver=(i+xlen)%nqubits.getValue();
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	}
	for(int i=0;i<nqubits.getValue();i++) {
	  Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	}
	for(int i=1;i<nqubits.getValue();i+=2) {
	  i_hor=(i+1)%xlen+(i/xlen)*xlen;
	  i_ver=(i+xlen)%nqubits.getValue();
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	}
      }
    }
    //Se aplica U^-1
    for(int t=0;t<numt.getValue();t++) {
      for(int it=0;it<num_trotter;it++) {
	for(int i=1;i<nqubits.getValue();i+=2) {
	  i_hor=(i+1)%xlen+(i/xlen)*xlen;
	  i_ver=(i+xlen)%nqubits.getValue();
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,-1*isin,l);
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,-1*isin,l);
	}
	for(int i=0;i<nqubits.getValue();i++) {
	  Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	}
	for(int i=0;i<nqubits.getValue();i+=2) {
	  i_hor=(i+1)%xlen+(i/xlen)*xlen;
	  i_ver=(i+xlen)%nqubits.getValue();
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,-1*isin,l);
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,-1*isin,l);
	}
      }
    }
    
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      finalstate(i)=std::complex<double>(R[i],I[i]);
    }
    cout<<itpp::norm(initialstate-finalstate)<<endl;
  } // }}}
  if (option=="correlation_measure") { // {{{
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double bx2,by2,bz2,kcos,ksin,icos,isin;
    itpp::vec b(3); b(0)=bx.getValue(); b(1)=by.getValue(); b(2)=bz.getValue();
    set_parameters(ising.getValue(),b,icos,isin,kcos,ksin,bx2,by2,bz2);
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
//     double res,norm;
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    cudaCheckError("sum_dx",1);
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
      
      //se aplica U^-1
      for(int t=0;t<n;t++) {
	for(int i=0;i<nqubits.getValue();i++) {
	  Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	  //cudaCheckError("kick",i);
	}
	for(int i=0;i<nqubits.getValue();i++) {
	  Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	  //cudaCheckError("ising",i);
	}
      }
      
      //se aplica la  U 
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
      for(int i=0;i<l;i++) {
	finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
      }
      res=std::norm(itpp::dot(itpp::conj(zerostate),finalstate));
      //std::cout<<itpp::dot(itpp::conj(zerostate),finalstate);
      cout<<sqrt(res)/nqubits.getValue()<<endl;
    }
  } // }}}
  if (option=="correlation_measure_test") { // {{{
    //this method computes fater correlations at the cost of having two states in global mem
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    double *zeroR=new double[l];
    double *zeroI=new double[l];
    double *dev_zeroR;      
    double *dev_zeroI;
    double *resR=new double[numt.getValue()];
    double *dev_resR;
    double *resI=new double[numt.getValue()];
    double *dev_resI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    hipMalloc((void**)&dev_zeroR,l*sizeof(double));     
    hipMalloc((void**)&dev_zeroI,l*sizeof(double));
    hipMalloc((void**)&dev_resR,numt.getValue()*sizeof(double));
    hipMalloc((void**)&dev_resI,numt.getValue()*sizeof(double));
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_zeroR,dev_zeroI,nqubits.getValue(),l);
    cudaCheckError("sum_dx",1);
    
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
      
      //se aplica U^-1
      for(int t=0;t<n;t++) {
	for(int i=0;i<nqubits.getValue();i++) {
	  Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	  //cudaCheckError("kick",i);
	}
	for(int i=0;i<nqubits.getValue();i++) {
	  Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	  //cudaCheckError("ising",i);
	}
      }
      
      //se aplica la  U 
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      timed_dot<<<numblocks,numthreads>>>(n,dev_zeroR,dev_zeroI,dev_sumdxR,dev_sumdxI,dev_resR,dev_resI,l);
      cudaCheckError("dot",0);
    }
    hipMemcpy(resR,dev_resR,numt.getValue()*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(resI,dev_resI,numt.getValue()*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<numt.getValue();i++) {
      cout<<sqrt(resR[i]*resR[i]+resI[i]*resI[i])/nqubits.getValue()<<endl;
    }
  } // }}}
  if (option=="fidelity_measure") { // {{{
    double *AR=new double[l];
    double *AI=new double[l];
    double *dev_AR;      
    double *dev_AI;
    hipMalloc((void**)&dev_AR,l*sizeof(double));     
    hipMalloc((void**)&dev_AI,l*sizeof(double));
    double delta=deltav.getValue();
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    double thetadel=delta;
    double bx2del=1.;
    double by2del=0.;
    double bz2del=0.;
    double kcosdel=cos(thetadel);
    double ksindel=sin(thetadel);
    itpp::cvec leftstate(l);
    itpp::cvec rightstate(l);
    for(int i=0;i<l;i++) {
      leftstate(i)=std::complex<double>(R[i],I[i]);
      rightstate(i)=std::complex<double>(R[i],I[i]);
    }
    cout<<std::norm(itpp::dot(itpp::conj(leftstate),rightstate))<<endl;
//     devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_AR,dev_AI);
    for(int n=0;n<numt.getValue();n++) {
      //se aplica la U
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	// cudaCheckError("ising",i);
      }
      
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      //Se aplica M_delta
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2del,by2del,bz2del,kcosdel,ksindel,l);
	//cudaCheckError("kick",i);
      }
      
      
      
      //Se aplica U^-1
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_AR,dev_AI,icos,isin,l);	  
	//cudaCheckError("kick",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_AR,dev_AI,bx2,by2,bz2,kcos,ksin,l);	  
	//cudaCheckError("ising",i);
      }
      
      
      
      hipMemcpy(AR,dev_AR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(AI,dev_AI,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
      for(int i=0;i<l;i++) {
	leftstate(i)=std::complex<double>(R[i],I[i]);
	rightstate(i)=std::complex<double>(AR[i],AI[i]);
      }
      cout<<std::norm(itpp::dot(itpp::conj(leftstate),rightstate))<<endl;
    }
  } // }}}
  if (option=="fidelity_measure2d") { // {{{
    int xlen=x.getValue();
//     int ylen=y.getValue();
    double *AR=new double[l];
    double *AI=new double[l];
    double *dev_AR;      
    double *dev_AI;
    hipMalloc((void**)&dev_AR,l*sizeof(double));     
    hipMalloc((void**)&dev_AI,l*sizeof(double));
    double delta=deltav.getValue();
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    double thetadel=delta;
    double bx2del=1.;
    double by2del=0.;
    double bz2del=0.;
    double kcosdel=cos(thetadel);
    double ksindel=sin(thetadel);
    int i_hor,i_ver;
    itpp::cvec leftstate(l);
    itpp::cvec rightstate(l);
    for(int i=0;i<l;i++) {
      leftstate(i)=std::complex<double>(R[i],I[i]);
      rightstate(i)=std::complex<double>(R[i],I[i]);
    }
    cout<<std::norm(itpp::dot(itpp::conj(leftstate),rightstate))<<endl;
//     devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_AR,dev_AI);
    for(int n=0;n<numt.getValue();n++) {
      //se aplica la U
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	// cudaCheckError("ising",i);
      }
      
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      //Se aplica M_delta
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2del,by2del,bz2del,kcosdel,ksindel,l);
	//cudaCheckError("kick",i);
      }
      
      
      
      //Se aplica U^-1
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_AR,dev_AI,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_AR,dev_AI,icos,isin,l);	  
	//cudaCheckError("kick",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_AR,dev_AI,bx2,by2,bz2,kcos,ksin,l);	  
	//cudaCheckError("ising",i);
      }
      
      
      
      hipMemcpy(AR,dev_AR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(AI,dev_AI,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
      for(int i=0;i<l;i++) {
	leftstate(i)=std::complex<double>(R[i],I[i]);
	rightstate(i)=std::complex<double>(AR[i],AI[i]);
      }
      cout<<std::norm(itpp::dot(itpp::conj(leftstate),rightstate))<<endl;
    }
  } // }}}
  if (option=="QFT") { // {{{
    for(int n=0;n<numt.getValue();n++) {
      for(int i=0;i<nqubits.getValue();i++) {
	QFT<<<numblocks,numthreads>>>(i,dev_R,dev_I,l,nqubits.getValue());
      }
    }
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      std::cout<<R[i]<<"  i"<<I[i]<<endl;
    }
  } // }}}
  if (option=="correlation_measure_carlos") { // {{{
    itpp::vec magnetic_field(3);
    magnetic_field(0)=bx.getValue();
    magnetic_field(1)=by.getValue();
    magnetic_field(2)=bz.getValue();
    int qubits = nqubits.getValue();
    int xlen=x.getValue();
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    cudaCheckError("sum_dx",1);
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
      //se aplica U^-1
      evcuda::apply_floquet2d(dev_sumdxR,dev_sumdxI, magnetic_field, ising.getValue() ,qubits, xlen);
      //se aplica U
      evcuda::apply_floquet2d(dev_R, dev_I, magnetic_field, ising.getValue() ,qubits, xlen);
      // se calcula el producto punto
      itppcuda::cuda2itpp(finalstate,dev_sumdxR, dev_sumdxI);
      res=std::norm(itpp::dot(itpp::conj(zerostate),finalstate));
      cout<<sqrt(res)/nqubits.getValue()<<endl;
    }
  } // }}}
  if (option=="correlation_measure2d") { // {{{
    int xlen=x.getValue();
//     int ylen=y.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));
    double bx2,by2,bz2,kcos,ksin,icos,isin;
    itpp::vec b(3); b(0)=bx.getValue(); b(1)=by.getValue(); b(2)=bz.getValue();
    set_parameters(ising.getValue(),b,icos,isin,kcos,ksin,bx2,by2,bz2);
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
//     double res,norm;
    int i_hor,i_ver;
    itpp::vec b_obs(3); b_obs(0)=1.; b_obs(1)=0.; b_obs(2)=0.;
    double cos_obs,sin_obs,bx_obs,by_obs,bz_obs;
    set_parameters(b_obs,cos_obs,sin_obs,bx_obs,by_obs,bz_obs);
    
    //sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
    //OBSERBABLE
	devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_inR,dev_inI);
	for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx_obs,by_obs,bz_obs,cos_obs,sin_obs,l);
	    //cudaCheckError("kick",i);
	  }
    
    //cudaCheckError("sum_dx",1);
    
    
    
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      //sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
      //OBSERBABLE
	devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_sumdxR,dev_sumdxI);
	for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,bx_obs,by_obs,bz_obs,cos_obs,sin_obs,l);
	    //cudaCheckError("kick",i);
	  }
      
      
      hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
      hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
      
      //se aplica U a in
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      //se aplica la  U 
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
     
      for(int i=0;i<l;i++) {
	finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	zerostate(i)=std::complex<double>(inR[i],inI[i]);
      }
      res=std::norm(itpp::dot(itpp::conj(zerostate),finalstate));
      cout<<sqrt(res)/nqubits.getValue()<<endl;
    }
  } // }}} 
  if (option=="correlation_obsz") { // {{{
    int xlen=x.getValue();
//     int ylen=y.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double kcos=cos(theta);
    double ksin=sin(theta);
    if(theta==0) {
      theta=1.;
    }
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
//     double res,norm;
    int i_hor,i_ver;
    
    sumsigma_z<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
    //cudaCheckError("sum_dx",1);
    
    
    
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      sumsigma_z<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
      
      hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
      hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
      
      //se aplica U a in
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      //se aplica la  U 
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
     
      for(int i=0;i<l;i++) {
	finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	zerostate(i)=std::complex<double>(inR[i],inI[i]);
      }
      res=std::norm(itpp::dot(itpp::conj(zerostate),finalstate));
      cout<<sqrt(res)/nqubits.getValue()<<endl;
    }
  } // }}}
  if (option=="correlation_obsy") { // {{{
    int xlen=x.getValue();
//     int ylen=y.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double kcos=cos(theta);
    double ksin=sin(theta);
    if(theta==0) {
      theta=1.;
    }
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
//     double res,norm;
    int i_hor,i_ver;
    
    sigma_xsigma_y<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,1,3,l);
    //cudaCheckError("sum_dx",1);
    
    
    
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      sigma_xsigma_y<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,1,3,l);
      
      hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
      hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
      
      //se aplica U a in
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
      //se aplica la  U 
      for(int i=0;i<nqubits.getValue();i++) {
	i_hor=(i+1)%xlen+(i/xlen)*xlen;
	i_ver=(i+xlen)%nqubits.getValue();
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	//cudaCheckError("kick",i);
      }
      
     
      for(int i=0;i<l;i++) {
	finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	zerostate(i)=std::complex<double>(inR[i],inI[i]);
      }
      res=std::norm(itpp::dot(itpp::conj(zerostate),finalstate));
      cout<<sqrt(res)/nqubits.getValue()<<endl;
    }
  } // }}}
  if (option=="color_map2d_no") { // {{{
    int xlen=x.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
    int i_hor,i_ver;
    double pass;
    int tgo,tback,cont;
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(double bxi=0.0;bxi<1.5;bxi+=0.05) {
      for(double bzi=0.0;bzi<1.5;bzi+=0.05) {
	pass=10.;
	cont=3;
	res=0.;
	double theta=sqrt((bxi*bxi)+(bzi*bzi));
	double kcos=cos(theta);
	double ksin=sin(theta);
	if(theta==0) {
	  theta=1.;
	}
	double bx2=bxi/theta;
	double by2=0;
	double bz2=bzi/theta;
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	tgo=70;
	tback=70;
	while(abs(pass-res)>0.001) {
	  // 	cout<<"pass "<<abs(pass-res)<<endl;
	  pass=res;
	  for(int n=0;n<tgo;n++) {
	    
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	      //cudaCheckError("ising",i);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	      //cudaCheckError("kick",i);
	    }
	  }
	  //se aplica M
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  //cudaCheckError("kick",00);
	  //se aplica U^-1
	  for(int n=0;n<tback;n++) {
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	      // 	    cudaCheckError("kick",i);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      //cout << i << " " << i_hor << " " << i_ver << endl;
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      //cudaCheckError("ising",i);
	    }
	  }
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  tgo=cont;
	  tback=70+cont;
	  cont++;
	}
	cout<< bxi << " " << bzi <<" "<<res<<" "<<cont-3<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev") { // {{{
    int xlen=x.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res;
    int i_hor,i_ver;
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(double bxi=0.0;bxi<=1.5;bxi+=0.01) {
      for(double bzi=0.0;bzi<=0;bzi+=0.05) {
	double theta=sqrt((bxi*bxi)+(bzi*bzi));
	double kcos=cos(theta);
	double ksin=sin(theta);
	if(theta==0) {
	  theta=1.;
	}
	double bx2=bxi/theta;
	double by2=0;
	double bz2=bzi/theta;
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	
	for(int n=0;n<70;n++) {
	  
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	}
	
	for(int in=0;in<30;in++) {
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  for(int back=0;back<71+in;back++) {
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	      // 	    cudaCheckError("kick",i);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      //cout << i << " " << i_hor << " " << i_ver << endl;
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      //cudaCheckError("ising",i);
	    }
	  }
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_fast") { // {{{
    int xlen=x.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res,bx2,by2,bz2,kcos,ksin,icos,isin,ising;
    int i_hor,i_ver;    
    
    itpp::vec b_obs(3); b_obs(0)=1.; b_obs(1)=0.; b_obs(2)=0.;
    double cos_obs,sin_obs,bx_obs,by_obs,bz_obs;
    set_parameters(b_obs,cos_obs,sin_obs,bx_obs,by_obs,bz_obs);
    double bxi,bzi;
    for(int bxii=0;bxii<=360;bxii+=1) {
      for(int bzii=0;bzii<=180;bzii+=1) {
	bxi=bxii*itpp::pi/720.;
	bzi=bzii*itpp::pi/720.;
	itpp::vec b(3); b(0)=bxi; b(1)=0.; b(2)=0.;
	ising=bzi;
	set_parameters(ising,b,icos,isin,kcos,ksin,bx2,by2,bz2);
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	
	//OBSERBABLE
	sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
// 	devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_inR,dev_inI);
// 	for(int i=0;i<nqubits.getValue();i++) {
// 	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx_obs,by_obs,bz_obs,cos_obs,sin_obs,l);
// 	    //cudaCheckError("kick",i);
// 	  }
// 	  
	
	for(int n=0;n<70;n++) {
	  
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_inR,dev_inI,icos,isin,l);
	    cudaCheckError("ising",i);
	  }
	  
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	}
	
	for(int in=0;in<30;in++) {
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  //OBSERVABLE
// 	  devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_sumdxR,dev_sumdxI);
// 	  for(int i=0;i<nqubits.getValue();i++) {
// 	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,bx_obs,by_obs,bz_obs,cos_obs,sin_obs,l);
// 	    //cudaCheckError("kick",i);
// 	  }
	 
	  
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_inR,dev_inI,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	    zerostate(i)=std::complex<double>(inR[i],inI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  //res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)));
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_block") { // {{{
    int xlen=x.getValue();
    
    itpp::cmat eigenvectors1=evcuda::invariant_vectors(nqubits.getValue(),x.getValue(),1,1,0);
	itpp::cmat eigenvectors2=evcuda::invariant_vectors(nqubits.getValue(),x.getValue(),1,2,0);

	int rcont1=eigenvectors1.rows();
	int rcont2=eigenvectors2.rows();
	
	
	itpp::cvec small_state=itppextmath::RandomState(rcont1);
	itpp::cvec state = itpp::transpose(eigenvectors1)*small_state;
	small_state=itppextmath::RandomState(rcont2);
	state=state+itpp::transpose(eigenvectors2)*small_state;
	state=state/itpp::norm(state);
	
	evcuda::itpp2cuda(state,dev_R,dev_I);
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res,bx2,by2,bz2,kcos,ksin,icos,isin,ising;
    int i_hor,i_ver;    
    
    itpp::vec b_obs(3); b_obs(0)=1./sqrt(3); b_obs(1)=1./sqrt(3); b_obs(2)=1./sqrt(3);
    double cos_obs,sin_obs,bx_obs,by_obs,bz_obs;
    set_parameters(b_obs,cos_obs,sin_obs,bx_obs,by_obs,bz_obs);
    
    for(double bxi=0.0;bxi<=itpp::pi/2;bxi+=itpp::pi/80) {
      for(double bzi=0.0;bzi<=itpp::pi/2;bzi+=itpp::pi/80) {
	itpp::vec b(3); b(0)=bxi; b(1)=0.; b(2)=0.;
	ising=bzi;
	set_parameters(ising,b,icos,isin,kcos,ksin,bx2,by2,bz2);
	evcuda::itpp2cuda(state,dev_R,dev_I);
	
	//OBSERBABLE
	sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
	//devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_inR,dev_inI);
// 	for(int i=0;i<nqubits.getValue();i++) {
// 	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx_obs,by_obs,bz_obs,cos_obs,sin_obs,l);
// 	    //cudaCheckError("kick",i);
// 	  }
	  
	
	for(int n=0;n<70;n++) {
	  
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //CHAIN
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_inR,dev_inI,icos,isin,l);
	    cudaCheckError("ising",i);
	  }
	  
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //CHAIN
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	}
	
	for(int in=0;in<1;in++) {
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  //OBSERVABLE
// 	  devcpy<<<numblocks,numthreads>>>(l,dev_R,dev_I,dev_sumdxR,dev_sumdxI);
// 	  for(int i=0;i<nqubits.getValue();i++) {
// 	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,bx_obs,by_obs,bz_obs,cos_obs,sin_obs,l);
// 	    //cudaCheckError("kick",i);
// 	  }
	 
	  
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //CHAIN
	    //Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_inR,dev_inI,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //CHAIN
	    Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	    zerostate(i)=std::complex<double>(inR[i],inI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_fast_obsz") { // {{{
    int xlen=x.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res,bx2,by2,bz2,kcos,ksin,ising,icos,isin;
    int i_hor,i_ver;    
    
    for(double bxi=0.;bxi<=3.2;bxi+=0.05) {
      for(double bzi=0.;bzi<=3.2;bzi+=0.05) {
	itpp::vec b(3); b(0)=bxi; b(1)=0.; b(2)=0.;
	ising=bzi;
	set_parameters(ising,b,icos,isin,kcos,ksin,bx2,by2,bz2);
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	sumsigma_z<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
	
	for(int n=0;n<70;n++) {
	  
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	}
	
	for(int in=0;in<30;in++) {
	  sumsigma_z<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	    zerostate(i)=std::complex<double>(inR[i],inI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_fast_obsy") { // {{{
    int xlen=x.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res,bx2,by2,bz2,kcos,ksin,ising,icos,isin;
    int i_hor,i_ver;    
    
    for(double bxi=0.;bxi<=3.2;bxi+=0.05) {
      for(double bzi=0.;bzi<=3.2;bzi+=0.05) {
	itpp::vec b(3); b(0)=bxi; b(1)=0.; b(2)=0.;
	ising=bzi;
	set_parameters(ising,b,icos,isin,kcos,ksin,bx2,by2,bz2);
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	sumsigma_y<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
	
	for(int n=0;n<70;n++) {
	  
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	}
	
	for(int in=0;in<30;in++) {
	  sumsigma_y<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	    zerostate(i)=std::complex<double>(inR[i],inI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_fast_in70") { // {{{
    int xlen=x.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(1);
     double res,bx2,by2,bz2,kcos,ksin,ising,icos,isin;  
    
    for(double bxi=0.0;bxi<=3.2;bxi+=0.05) {
      for(double bzi=0.0;bzi<=3.2;bzi+=0.05) {
	itpp::vec b(3); b(0)=bxi; b(1)=0.; b(2)=0.;
	ising=bzi;
	set_parameters(ising,b,icos,isin,kcos,ksin,bx2,by2,bz2);
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
	
	for(int n=0;n<10;n++) {
	  
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_inR,dev_inI,icos,isin,l);
	//cudaCheckError("ising",i);
      }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	}
	
	for(int in=0;in<1;in++) {
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
	  
	  //se aplica U a in
	  for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_inR,dev_inI,icos,isin,l);
	//cudaCheckError("ising",i);
      }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  //se aplica la  U 
	  for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  
	  
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	    zerostate(i)=std::complex<double>(inR[i],inI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)/(double)nqubits.getValue()));
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev__trotter1g_fast") { // {{{
    int xlen=x.getValue();
    int num_trotter=trotternum.getValue();
    double delta=1./num_trotter;
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos(delta*ising.getValue());
    double isin=sin(delta*ising.getValue());
    double *inR=new double[l];
    double *inI=new double[l];
    double *dev_inR;      
    double *dev_inI;
    hipMalloc((void**)&dev_inR,l*sizeof(double));     
    hipMalloc((void**)&dev_inI,l*sizeof(double));    
    
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res;
    int i_hor,i_ver;    
    
    for(double bxi=0.0;bxi<=1.5;bxi+=0.05) {
      for(double bzi=0.0;bzi<=1.5;bzi+=0.05) {
	double theta=delta*sqrt((bxi*bxi)+(bzi*bzi));
	double kcos=cos(theta);
	double ksin=sin(theta);
	if(theta==0) {
	  theta=1.;
	}
	double bx2=delta*bxi/theta;
	double by2=0;
	double bz2=delta*bzi/theta;
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_inR,dev_inI,nqubits.getValue(),l);
	
	for(int n=0;n<70;n++) {
	  
	  
	  //se aplica U a in
	  for(int trot=0;trot<num_trotter;trot++) {
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  }
	  
	  //se aplica la  U 
	  for(int trot=0;trot<num_trotter;trot++) {
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  }
	}
	
	for(int in=0;in<30;in++) {
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inR,dev_inR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(inI,dev_inI,l*sizeof(double),hipMemcpyDeviceToHost);   
	  
	  //se aplica U a in
	  for(int trot=0;trot<num_trotter;trot++) {
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_inR,dev_inI,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_inR,dev_inI,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_inR,dev_inI,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  }
	  
	  //se aplica la  U 
	  for(int trot=0;trot<num_trotter;trot++) {
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    //cudaCheckError("ising",i);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    //cudaCheckError("kick",i);
	  }
	  }
	  
	  
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	    zerostate(i)=std::complex<double>(inR[i],inI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_trotter2g") { // {{{
    int xlen=x.getValue();
    int num_trotter=trotternum.getValue();
    double delta=1./num_trotter;
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos((delta/2)*ising.getValue());
    double isin=sin((delta/2)*ising.getValue());
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res;
    int i_hor,i_ver;
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(double bxi=beginx.getValue();bxi<=1.5;bxi+=0.05) {
      for(double bzi=0.0;bzi<=1.5;bzi+=0.05) {
	double theta=delta*sqrt((bxi*bxi)+(bzi*bzi));
	double kcos=cos(theta);
	double ksin=sin(theta);
	if(theta==0) {
	  theta=1.;
	}
	double bx2=delta*bxi/theta;
	double by2=0;
	double bz2=delta*bzi/theta;
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	
	for(int n=0;n<70;n++) {
	  
	  for(int it=0;it<num_trotter;it++) {
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    }
	  }
	}
	
	for(int in=0;in<30;in++) {
	  for(int it=0;it<num_trotter;it++) {
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    }
	  }
	  
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  for(int back=0;back<71+in;back++) {
	    for(int it=0;it<num_trotter;it++) {
	      for(int i=0;i<nqubits.getValue();i++) {
		i_hor=(i+1)%xlen+(i/xlen)*xlen;
		i_ver=(i+xlen)%nqubits.getValue();
		Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
		Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      }
	      for(int i=0;i<nqubits.getValue();i++) {
		Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	      }
	      for(int i=0;i<nqubits.getValue();i++) {
		i_hor=(i+1)%xlen+(i/xlen)*xlen;
		i_ver=(i+xlen)%nqubits.getValue();
		Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
		Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      }
	    }
	  }
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="color_map2d_stdev_trotter1g") { // {{{
    int xlen=x.getValue();
    int num_trotter=trotternum.getValue();
    double delta=1./num_trotter;
    cout<<"delta " << delta<<endl;
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos((delta)*ising.getValue());
    double isin=sin((delta)*ising.getValue());
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    itpp::vec correlations(30);
    double res;
    int i_hor,i_ver;
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(double bxi=beginx.getValue();bxi<1.5;bxi+=0.05) {
      for(double bzi=0.;bzi<1.5;bzi+=0.05) {
	double theta=delta*sqrt((bxi*bxi)+(bzi*bzi));
	double kcos=cos(theta);
	double ksin=sin(theta);
	if(theta==0) {
	  theta=1.;
	}
	double bx2=delta*bxi/theta;
	double by2=0;
	double bz2=delta*bzi/theta;
	hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
	
	for(int n=0;n<70;n++) {
	  
	  for(int it=0;it<num_trotter;it++) {
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    }
	  }
	}
	
	for(int in=0;in<30;in++) {
	  for(int it=0;it<num_trotter;it++) {
	    for(int i=0;i<nqubits.getValue();i++) {
	      i_hor=(i+1)%xlen+(i/xlen)*xlen;
	      i_ver=(i+xlen)%nqubits.getValue();
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	      Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	    }
	    for(int i=0;i<nqubits.getValue();i++) {
	      Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	    }
	  }
	  
	  sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
	  
	  for(int back=0;back<71+in;back++) {
	    for(int it=0;it<num_trotter;it++) {
	      for(int i=0;i<nqubits.getValue();i++) {
		Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	      }
	      for(int i=0;i<nqubits.getValue();i++) {
		i_hor=(i+1)%xlen+(i/xlen)*xlen;
		i_ver=(i+xlen)%nqubits.getValue();
		Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
		Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	      }         
	    }
	  }
	  hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
	  hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
	  for(int i=0;i<l;i++) {
	    finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
	  }
	  res=sqrt(std::norm(itpp::dot(itpp::conj(zerostate),finalstate)))/nqubits.getValue();
	  correlations(in)=res;
	}
	cout<<bxi<<" "<<bzi<<" "<<itpp::mean(correlations)<<" "<<std::sqrt(itpp::variance(correlations))<<endl;
      }
    }
  } // }}}
  if (option=="test_2d_grid") { // {{{
    int xlen=x.getValue();
//     int ylen=y.getValue();
    
    
    //cout<<square<<endl;
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    
    for(int n=0;n<numt.getValue();n++) { 
      for(int i=0;i<nqubits.getValue();i++) {
	int i_hor=(i+1)%xlen+(i/xlen)*xlen;
	int i_ver=(i+xlen)%nqubits.getValue();
	cout << i << " " << i_hor << " " << i_ver << endl;
	Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	//cudaCheckError("ising",i);
      }
    }
    
    hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
    hipFree(dev_R);
    hipFree(dev_I);
    for(int i=0;i<l;i++) {
      std::cout<<R[i]<<"  i"<<I[i]<<endl;
    }
  }  // }}}
  if (option=="exp_lattice") { // {{{
    //nqubits debe corresponder a un cuadro
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(pow(bx.getValue(),2)+pow(by.getValue(),2)+pow(bz.getValue(),2));
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    double *dotR=new double[nqubits.getValue()];
    double *dotI=new double[nqubits.getValue()];
    double *dev_dotR;      
    double *dev_dotI;
    hipMalloc((void**)&dev_dotR,nqubits.getValue()*sizeof(double));     
    hipMalloc((void**)&dev_dotI,nqubits.getValue()*sizeof(double));
    //cout<<div<<endl;
    
    for(int t=0;t<numt.getValue();t++) {
      for(int i=0;i<nqubits.getValue();i++) {
	dotR[i]=0;
	dotI[i]=0;
      }
      cudaSafeCall(hipMemcpy(dev_dotR,dotR,nqubits.getValue()*sizeof(double),hipMemcpyHostToDevice),"hipMalloc",1);
      cudaSafeCall(hipMemcpy(dev_dotI,dotI,nqubits.getValue()*sizeof(double),hipMemcpyHostToDevice),"hipMalloc",2);
      for(int i=0;i<nqubits.getValue();i++) {
	// 	dot_2<<<numblocks,numthreads>>>(1,1,i,dev_R,dev_I,dev_dotR,dev_dotI,l);
	//hipDeviceSynchronize();
	cudaCheckError("dot",i);
      }
      cudaSafeCall(hipMemcpy(dotR,dev_dotR,nqubits.getValue()*sizeof(double),hipMemcpyDeviceToHost),"hipMalloc",3);
      cudaSafeCall(hipMemcpy(dotI,dev_dotI,nqubits.getValue()*sizeof(double),hipMemcpyDeviceToHost),"hipMalloc",4);
      for(int i=0;i<nqubits.getValue();i++) {
	cout<<dotR[i]<<" ";
      }
      cout<<endl;
      for(int is=0;is<nqubits.getValue();is++) {
	//Ui_kernel<<<numblocks,numthreads>>>(is,(is+1)%5,dev_R,dev_I,icos,isin,l);
	Ui_kernel<<<numblocks,numthreads>>>(is,(is+5)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	cudaCheckError("ising",is);
      }
      for(int ki=0;ki<nqubits.getValue();ki++) {
	Uk_kernel<<<numblocks,numthreads>>>(ki,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	cudaCheckError("kick",ki);
      }
    }
    hipFree(dev_dotR);
    hipFree(dev_dotI); 
  } // }}}
  if (option=="test_densmat") { // {{{
    int ndens=pow(2,numbits(whichq.getValue()));
    double *densR=new double[ndens*ndens];
    double *densI=new double[ndens*ndens];
    double *dev_densR;      
    double *dev_densI;
    hipMalloc((void**)&dev_densR,ndens*ndens*sizeof(double));     
    hipMalloc((void**)&dev_densI,ndens*ndens*sizeof(double));
    for(int i=0;i<ndens*ndens;i++) {
      densR[i]=0;
      densI[i]=0;
    }
    cudaSafeCall(hipMemcpy(dev_densR,densR,ndens*ndens*sizeof(double),hipMemcpyHostToDevice),"hipMemcpy",1);
    cudaSafeCall(hipMemcpy(dev_densI,densI,ndens*ndens*sizeof(double),hipMemcpyHostToDevice),"hipMemcpy",2);
    int div=choosediv(nqubits.getValue());
    //     int blockdivdens,threaddivdens;
    //     choosedivdens(l,blockdivdens,threaddivdens);
    cout<<numblocks<<"  "<<numthreads/ndens<<endl;
    for(int i=0;i<ndens;i++) {
      for(int j=0;j<ndens;j++) {
	density_matrix<<<numblocks,numthreads/ndens>>>(whichq.getValue(),ndens,i,j,dev_R,dev_I,dev_densR,dev_densI,l/ndens);
	cudaCheckError("dot",i+j);
      }
    }
    cudaSafeCall(hipMemcpy(densR,dev_densR,ndens*ndens*sizeof(double),hipMemcpyDeviceToHost),"hipMemcpy",3);
    cudaSafeCall(hipMemcpy(densI,dev_densI,ndens*ndens*sizeof(double),hipMemcpyDeviceToHost),"hipMemcpy",4);
    for(int i=0;i<ndens;i++) {
      for(int j=0;j<ndens;j++) {
	cout<<densR[(ndens*i)+j]<<" i"<<densI[(ndens*i)+j]<<" ";
     }
      cout<<endl;
    }
    hipFree(dev_densR);
    hipFree(dev_densI);
  } // }}}
  if (option=="exp_cadena_densmat") { // {{{
    double icos=cos(ising.getValue());
    double isin=sin(ising.getValue());
    double theta=sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=bx.getValue()/theta;
    double by2=by.getValue()/theta;
    double bz2=bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    int ndens=pow(2,numbits(whichq.getValue()));
    double *densR=new double[ndens*ndens];
    double *densI=new double[ndens*ndens];
    double *dev_densR;      
    double *dev_densI;
    hipMalloc((void**)&dev_densR,ndens*ndens*sizeof(double));     
    hipMalloc((void**)&dev_densI,ndens*ndens*sizeof(double));
    itpp::cmat dens(2,2);
    // which temp para sacar matdens todos
    //for(int n=0;n<nqubits.getValue();n++) {
    for(int n=0;n<numt.getValue();n++) {
      for(int qus=0;qus<nqubits.getValue();qus++) {
	int whichtemp=pow(2,qus);
	for(int i=0;i<ndens*ndens;i++) {
	  densR[i]=0;
	  densI[i]=0;
	}
	cudaSafeCall(hipMemcpy(dev_densR,densR,ndens*ndens*sizeof(double),hipMemcpyHostToDevice),"hipMemcpy",1);
	cudaSafeCall(hipMemcpy(dev_densI,densI,ndens*ndens*sizeof(double),hipMemcpyHostToDevice),"hipMemcpy",2);
	for(int i=0;i<ndens;i++) {
	  for(int j=0;j<ndens;j++) {
	    //whichtemp aqui
	    density_matrix<<<numblocks,numthreads/ndens>>>(whichtemp,ndens,i,j,dev_R,dev_I,dev_densR,dev_densI,l/ndens);
	    cudaCheckError("dot",i+j);
	  }
	}
	cudaSafeCall(hipMemcpy(densR,dev_densR,ndens*ndens*sizeof(double),hipMemcpyDeviceToHost),"hipMemcpy",3);
	cudaSafeCall(hipMemcpy(densI,dev_densI,ndens*ndens*sizeof(double),hipMemcpyDeviceToHost),"hipMemcpy",4);
	for(int i=0;i<ndens;i++) {
	  for(int j=0;j<ndens;j++) {
	    dens(i,j)=std::complex<double>(densR[(ndens*i)+j],densI[(ndens*i)+j]);
	  }
	}
	cout<<real(itpp::trace(dens*itppextmath::sigma(3)))<<endl;
	//       cout<<"-----------------otro qubit-----------------------"<<endl;
      }
      for(int i=0;i<nqubits.getValue();i++) {
	Ui_kernel<<<numblocks,numthreads>>>(i,(i+1)%nqubits.getValue(),dev_R,dev_I,icos,isin,l);
	cudaCheckError("ising",i);
      }
      
      for(int i=0;i<nqubits.getValue();i++) {
	Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	cudaCheckError("ising",i);
      }
      //cout<<endl;
    }
    hipFree(dev_densR);
    hipFree(dev_densI);
  } // }}}
  if(option=="correlation_measure2d_trotter") { // {{{
    int num_trotter=trotternum.getValue();
    double delta=1./num_trotter;
    int xlen=x.getValue();
//     int ylen=y.getValue();
    //     cout<<square<<endl;
    double *sumdxR=new double[l];
    double *sumdxI=new double[l];
    double *dev_sumdxR;      
    double *dev_sumdxI;
    hipMalloc((void**)&dev_sumdxR,l*sizeof(double));     
    hipMalloc((void**)&dev_sumdxI,l*sizeof(double));
    double icos=cos((delta/2)*ising.getValue());
    double isin=sin((delta/2)*ising.getValue());
    double theta=(delta)*sqrt(bx.getValue()*bx.getValue()+by.getValue()*by.getValue()+bz.getValue()*bz.getValue());
    double bx2=delta*bx.getValue()/theta;
    double by2=delta*by.getValue()/theta;
    double bz2=delta*bz.getValue()/theta;
    double kcos=cos(theta);
    double ksin=sin(theta);
    itpp::cvec finalstate(l);
    itpp::cvec zerostate(l);
    double res;
//     double res,norm;
    int i_hor,i_ver;
    
    sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
    cudaCheckError("sum_dx",1);
    
    hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
    for(int i=0;i<l;i++) {
      zerostate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
    }
    
    for(int n=0;n<numt.getValue();n++) {
      //se aplica M
      sumsigma_x<<<numblocks,numthreads>>>(dev_R,dev_I,dev_sumdxR,dev_sumdxI,nqubits.getValue(),l);
      
      //se aplica U^-1
      for(int t=0;t<n;t++) {
	for(int it=0;it<num_trotter;it++) {
	  for(int i=1;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    Uk_kernel<<<numblocks,numthreads>>>(i,dev_sumdxR,dev_sumdxI,-1*bx2,-1*by2,-1*bz2,kcos,ksin,l);
	  }
	  for(int i=0;i<nqubits.getValue();i++) {
	    i_hor=(i+1)%xlen+(i/xlen)*xlen;
	    i_ver=(i+xlen)%nqubits.getValue();
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	    Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_sumdxR,dev_sumdxI,icos,-1*isin,l);
	  }
	}
      }
      
      //se aplica la  U 
      for(int it=0;it<num_trotter;it++) {
	for(int i=0;i<nqubits.getValue();i++) {
	  i_hor=(i+1)%xlen+(i/xlen)*xlen;
	  i_ver=(i+xlen)%nqubits.getValue();
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	}
	for(int i=0;i<nqubits.getValue();i++) {
	  Uk_kernel<<<numblocks,numthreads>>>(i,dev_R,dev_I,bx2,by2,bz2,kcos,ksin,l);
	}
	for(int i=1;i<nqubits.getValue();i++) {
	  i_hor=(i+1)%xlen+(i/xlen)*xlen;
	  i_ver=(i+xlen)%nqubits.getValue();
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_hor,dev_R,dev_I,icos,isin,l);
	  Ui_kernel<<<numblocks,numthreads>>>(i,i_ver,dev_R,dev_I,icos,isin,l);
	}
      }
      
      hipMemcpy(sumdxR,dev_sumdxR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(sumdxI,dev_sumdxI,l*sizeof(double),hipMemcpyDeviceToHost);
      for(int i=0;i<l;i++) {
	finalstate(i)=std::complex<double>(sumdxR[i],sumdxI[i]);
      }
      res=std::norm(itpp::dot(itpp::conj(zerostate),finalstate));
      cout<<sqrt(res)/nqubits.getValue()<<endl;
    }
  } // }}}
  if (option=="test_proyector_big") { // {{{
    itpp::cvec proyectado(l);
    double *rotR=new double[l];
    double *rotI=new double[l];
    double *dev_rotR;      
    double *dev_rotI;
    hipMalloc((void**)&dev_rotR,l*sizeof(double));     
    hipMalloc((void**)&dev_rotI,l*sizeof(double));
    
    for(int i=1;i<x.getValue();i++) {
      vertical_rotation<<<numblocks,numthreads>>>(dev_R,dev_I,dev_rotR,dev_rotI,x.getValue(),nqubits.getValue(),l,i);
      hipMemcpy(rotR,dev_rotR,l*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(rotI,dev_rotI,l*sizeof(double),hipMemcpyDeviceToHost);
      for(int j=0;j<l;j++) {
	R[j]=R[j]+cos(2*itpp::pi*km.getValue()*i/x.getValue())*rotR[j]-sin(2*itpp::pi*km.getValue()*i/x.getValue())*rotI[j];
	I[j]=I[j]+sin(2*itpp::pi*km.getValue()*i/x.getValue())*rotR[j]+cos(2*itpp::pi*km.getValue()*i/x.getValue())*rotI[j];
      }
    }
    
    hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
    
    vertical_rotation<<<numblocks,numthreads>>>(dev_R,dev_I,dev_rotR,dev_rotI,x.getValue(),nqubits.getValue(),l);
    hipMemcpy(rotR,dev_rotR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(rotI,dev_rotI,l*sizeof(double),hipMemcpyDeviceToHost);
    
    std::complex<double> fase=std::exp(std::complex<double>(0,-2*itpp::pi*km.getValue()/x.getValue()));    
    
    for(int i=0;i<l;i++) {
      //cout<<std::complex<double>(rotR[i],rotI[i])/std::complex<double>(R[i],I[i])<<endl;
      proyectado(i)=fase*(std::complex<double>(R[i],I[i]))-std::complex<double>(rotR[i],rotI[i]);
      //cout<<"---->"<<proyectado(i)<<endl;
    }
    cout<<"-----------------"<<endl;
    cout<<itpp::norm(proyectado)<<endl;
    //cout<<fase<<endl;
    
    
  } // }}}
  if (option=="test_proyector") { // {{{
    itpp::cvec proyectado(l);
    double *rotR=new double[l];
    double *rotI=new double[l];
    double *dev_rotR;      
    double *dev_rotI;
    hipMalloc((void**)&dev_rotR,l*sizeof(double));     
    hipMalloc((void**)&dev_rotI,l*sizeof(double));
    itpp::cvec vector(l);
    for(int i=0;i<l;i++) {
      vector(i)=std::complex<double>(R[i],I[i]);
    }
    vertical_proyector<<<numblocks,numthreads>>>(dev_R,dev_I,dev_rotR,dev_rotI,x.getValue(),nqubits.getValue(),l,km.getValue());    
    hipMemcpy(rotR,dev_rotR,l*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(rotI,dev_rotI,l*sizeof(double),hipMemcpyDeviceToHost);  
    for(int i=0;i<l;i++) {
      proyectado(i)=std::complex<double>(rotR[i],rotI[i]);
    }
    
    
    evcuda::proyector_vertical_itpp(vector,dev_R,dev_I,dev_rotR,dev_rotI,x.getValue(),km.getValue());
    
    cout<< proyectado<<endl;
    cout<<vector<<endl;
    cout<<itpp::norm(proyectado-vector)<<endl;
    itpp::cvec rotado=proyectado;
    itpp::cvec rotado2=vector;
    int nx = x.getValue();
    evcuda::apply_vertical_rotation_itpp(rotado,dev_R,dev_I,dev_rotR,dev_rotI,nx); 
    evcuda::apply_vertical_rotation_itpp(rotado2,dev_R,dev_I,dev_rotR,dev_rotI,nx); 
    
    double error = abs(itppextmath::proportionality_test(rotado,proyectado));
    double error2 = abs(itppextmath::proportionality_test(proyectado,vector));
    
    cout << "Error en la proporcionalidad es del cuda "  << error << endl;
    cout << "Error en la proporcionalidad es del normal-- "  << error2 << endl;
    //cout<<proyectado<<endl;
    //cout<<vector<<endl;
  } // }}}
  if (option=="assemble_matrix") { // {{{
    double *rotR=new double[l];
    double *rotI=new double[l];
    double *dev_rotR;      
    double *dev_rotI;
    hipMalloc((void**)&dev_rotR,l*sizeof(double));     
    hipMalloc((void**)&dev_rotI,l*sizeof(double)); 
    int *A=new int[l];
    for(int i=0;i<l;i++) {
      A[i]=2;
    }
    find_states_horizontal(A,nqubits.getValue(),x.getValue(),km.getValue(),l);
    int cont=0;
    for(int i=0;i<l;i++) {
      cont+=A[i];
    }
    itpp::cmat eigenvectors(cont,cont);
    for(int vec=0;vec<cont;vec++) {
      int flag=0;
      for(int i=0;i<l;i++) {
	if(A[i]=1 && flag==0) {
	  R[i]=1.;
	  flag=1;
	  A[i]=0;
	}
	else {
	  R[i]=0.;
	}
	I[i]=0;
      }
      hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
      hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);
      for(int i=1;i<x.getValue();i++) {
	horizontal_rotation<<<numblocks,numthreads>>>(dev_R,dev_I,dev_rotR,dev_rotI,x.getValue(),nqubits.getValue(),l,i);
	hipMemcpy(rotR,dev_rotR,l*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(rotI,dev_rotI,l*sizeof(double),hipMemcpyDeviceToHost);
	for(int j=0;j<l;j++) {
	  R[j]=R[j]+cos(2*itpp::pi*km.getValue()*i/x.getValue())*rotR[j]-sin(2*itpp::pi*km.getValue()*i/x.getValue())*rotI[j];
	  I[j]=I[j]+sin(2*itpp::pi*km.getValue()*i/x.getValue())*rotR[j]+cos(2*itpp::pi*km.getValue()*i/x.getValue())*rotI[j];
	}
      }
      
      
      for(int i=0;i<l;i++) {
	eigenvectors(vec,i)=std::complex<double>(R[i],I[i]);
      }
    }
  } // }}}
  hipFree(dev_R);
  hipFree(dev_I);
  // {{{ Final report
  if(!no_general_report.getValue()){
    error += system("echo \\#terminando:    $(date)");
  }
  // }}}
  return 0;
}



