#include "hip/hip_runtime.h"
#include "cu_complex.cu"

__global__ void dot_split(int i,int j,int split,double R[],double I[],double parR[],double parI[],int l) {
  int index=threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ Complex cache[512];
  int cacheindex=threadIdx.x;
  
  Complex temp;
  while(index<l/split) {
    Complex a=Complex(R[(l/split)*i+index],I[(l/split)*i+index]);
    Complex b=Complex(R[(l/split)*j+index],(-1)*I[(l/split)*j+index]);
    a=(a*b);
    temp= temp+a;
    index +=blockDim.x*gridDim.x;
  }
  
  cache[cacheindex]=temp;
  __syncthreads();
  
  int n = blockDim.x/2;
  while (n != 0) {
    if (cacheindex < n)
      cache[cacheindex] = cache[cacheindex] + cache[cacheindex + n];
    __syncthreads();
    n /= 2;
  }
  if (cacheindex==0) {
    parR[blockIdx.x]=cache[0].real;
    parI[blockIdx.x]=cache[0].imag;
  }
}

__global__ void dot_1(int i,int j,int q,double R[],double I[],double parR[],double parI[],int l) {
  int index=threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ Complex cache[512];
   int cacheindex=threadIdx.x;
  
  
  Complex temp=Complex(0,0);
  while (((index/__float2int_rz(powf(2,q)))%2==0) && (index<l)) {
    int i2=__float2int_rz(powf(2,q));
    Complex a=Complex(R[index+i*i2],I[index+i*i2]);
    Complex b=Complex(R[index+j*i2],(-1)*I[index+j*i2]);
    a=(a*b);
    temp= temp+a;
    index +=blockDim.x*gridDim.x;
  }
  
  cache[cacheindex]=temp;
  __syncthreads();
  
  int n = blockDim.x/2;
  while (n != 0) {
    if (cacheindex < n)
      cache[cacheindex] = cache[cacheindex] + cache[cacheindex + n];
    __syncthreads();
    n /= 2;
  }
  if (cacheindex==0) {
    parR[blockIdx.x]=cache[0].real;
    parI[blockIdx.x]=cache[0].imag;
  }
}


__global__ void dot_2(int i,int j,int q,double R[],double I[],double dotR[],double dotI[],int l) {
  int index=threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ Complex cache[512];
   int cacheindex=threadIdx.x;
  
  
  Complex temp=Complex(0,0);
  while (index<l) {
    if ((index/__float2int_rz(__powf(2,q)))%2==0) {
      int i2=__float2int_rz(powf(2,q));
      Complex a=Complex(R[index+i*i2],I[index+i*i2]);
      Complex b=Complex(R[index+j*i2],(-1)*I[index+j*i2]);
      a=(a*b);
      temp= temp+a;
    }
    index +=blockDim.x*gridDim.x;
  }
  
  cache[cacheindex]=temp;
  __syncthreads();
  
  int n = blockDim.x/2;
  while (n != 0) {
    if (cacheindex < n)
      cache[cacheindex] = cache[cacheindex] + cache[cacheindex + n];
    __syncthreads();
    n /= 2;
  }
  if (cacheindex==0) {
    matomicAdd(&dotR[q],cache[0].real);
    matomicAdd(&dotI[q],cache[0].imag);
  }
}


__global__ void density_matrix2(int which,int ndim,double R[],double I[],double densR[],double densI[],int l) {
  int index=threadIdx.x+blockIdx.x*blockDim.x;
  int i=0;
  int res,cont,i2;
  while (index<l) {
    res=0;
    cont=0;
    remove_zeros(which,index,res);
    do {
      if (i|which==which) {
	i2=i-index&which;
	Complex b=Complex(R[i2],I[i2]);
	Complex a=Complex(R[index],I[index])*b;	
	matomicAdd(&densR[res*ndim+cont],a.real);
	matomicAdd(&densI[res*ndim+cont],a.imag);
      }
      i++;
      cont++;
    } while (i!=which);
    index +=blockDim.x*gridDim.x;
  }
}

