//DIFERENTE A EVCUDA EN CUDA.GIT
#ifndef EVCUDA
# define EVCUDA
# include "tools.cpp"
# include "cuda_functions.cu"
namespace evcuda{ 

void cmalloc(double** dev_R,double** dev_I,int l) {
  double *source_R;
  double *source_I;
  hipMalloc((void**)&source_R,l*sizeof(double));
  hipMalloc((void**)&source_I,l*sizeof(double));
  *dev_R=source_R;
  *dev_I=source_I;
  }


void itpp2cuda_malloc(itpp::cvec& state,double** dev_R,double** dev_I) { 
  int l=state.size();
  double *R=new double[l];
  double *I=new double[l];
  double *source_R;
  double *source_I;
  for(int i=0;i<l;i++) {
    R[i]=real(state(i));
    I[i]=imag(state(i));
    }
  
//   hipMalloc((void**)&source_R,l*sizeof(double));
//   hipMalloc((void**)&source_I,l*sizeof(double));
    
  cudaSafeCall(hipMalloc((void**)&source_R,l*sizeof(double)),"malloc",0);
  cudaSafeCall(hipMalloc((void**)&source_I,l*sizeof(double)),"malloc",0);
  
  //cout<<source_R<<" "<<source_I<<endl;

  hipMemcpy(source_R,R,l*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(source_I,I,l*sizeof(double),hipMemcpyHostToDevice);

  //cout<<dev_R<<" "<<dev_I<<endl;
  *dev_R=source_R;
  *dev_I=source_I;

  delete[] R;
  delete[] I;
  }
void itpp2cuda(itpp::cvec& state,double* dev_R,double* dev_I) { 
  int l=state.size();
  double *R=new double[l];
  double *I=new double[l];
  for(int i=0;i<l;i++) {
    R[i]=real(state(i));
    I[i]=imag(state(i));
    }

  hipMemcpy(dev_R,R,l*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_I,I,l*sizeof(double),hipMemcpyHostToDevice);

  delete[] R;
  delete[] I;
  }
void cuda2itpp(itpp::cvec& state,double* dev_R,double* dev_I) { 
  int l=state.size();
  double *R=new double[l];
  double *I=new double[l];
  //cout<<dev_R<<" "<<dev_I<<endl;

  hipMemcpy(R,dev_R,l*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(I,dev_I,l*sizeof(double),hipMemcpyDeviceToHost);
  for(int i=0;i<l;i++) {
    state(i)=std::complex<double>(R[i],I[i]);
    }

  delete[] R;
  delete[] I;
  }

  

} 
#endif                                                    // EVCUDA
