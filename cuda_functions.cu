#include "hip/hip_runtime.h"
#include "cu_complex.cu"

#ifndef CUDAFUNCTS
#define CUDAFUNCTS

__device__ double normed;


__device__ long long int ipow(int base,int exp) {
  int i=exp;
  long long int res=1;
  while (i>0) {
    res*=base;
    i--;
  }
  return res;
}

__device__ long long int bit_reflection_h(int index,int nqubits) {
  long long int centro = (nqubits/2 + (nqubits%2) - 1);
  long long int j=1;
  long long int res=0;
  for(int i=0;i<nqubits;i++) {
    if((j&index)!=0) {
      if(i<=centro) {
        res += j<<(2*(centro-i) + (1-nqubits%2));
      }
      else {
	res += j>>(2*(i-centro) - (1-nqubits%2));
      }
    }
    j=j<<1;
  }
  return res;
}

__device__ long long int bit_rotation_h(int index,int x,int nqubits) {
  long long int temp,comp,a,b,res=0;
  for(int i=0;i<nqubits;i+=x) {
    temp=0,comp=0;
    for(int j=0;j<x;j++) {
      temp+=index&ipow(2,i+j);
      comp+=ipow(2,i+j);
    }
    a=(temp<<1)&comp;
    b=(temp>>(x-1))&comp;
    res+= a+b;
  }
  return res;
}


__device__ long long int bit_rotation_v(int index,int x,int nqubits) {
  long long int temp,comp,a,b,res=0;
  for(int i=0;i<x;i++) {
    temp=0,comp=0;
    for(int j=0;j<nqubits;j+=x) {
      temp+=index&ipow(2,i+j);
      comp+=ipow(2,i+j);
    }
    a=(temp<<x)&comp;
    b=(temp>>(nqubits-x))&comp;
    res+=a+b;
  }
  return res;
}


__device__  long long int trans(int index,int which,int where) {
  int i=1,cont=1,a,res=0;
  long long int warped=index;
  do { 
    if((i&which)==i) {
      a=(cont&where)/cont;
      cont*=2;
      res+=i*a;
      warped=warped<<1;
    } 
    if((i&warped)==i) {
      res+=i;
      warped-=i;
    }
    i*=2;
  } while(i<=warped || i<=which);
  return res;
}



__device__ void remove_zeros(int which,int in,int res) {
  int i=0;
  int cont=0;
  do {
    if (i|which==which) {
      if (i|in==in) {
	res+=ipow(2,cont);
      }
      cont++;
    }
    i++;
  } while (i!=which);
}

__global__ void vertical_rotation(double R[],double I[],double rotR[],double rotI[],int x,int nqubits,int l,int n=1) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  while (index<l) {
    index2=index;
    for(int i=0;i<n;i++) {
      index2=bit_rotation_v(index2,x,nqubits);
    }
    rotR[index2]=R[index];
    rotI[index2]=I[index];
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void horizontal_rotation(double R[],double I[],double rotR[],double rotI[],int x,int nqubits,int l,int n=1) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  while (index<l) {
    index2=index;
    for(int i=0;i<n;i++) {
      index2=bit_rotation_h(index2,x,nqubits);
    }
    rotR[index2]=R[index];
    rotI[index2]=I[index];
    index +=blockDim.x*gridDim.x;
  }
}





__global__ void horizontal_proyector(double R[],double I[],double rotR[],double rotI[],int x,int nqubits,int l,int k) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  while (index<l) {
    index2=index;
    rotR[index]=R[index];
    rotI[index]=I[index];
    for(int i=1;i<x;i++) {
      index2=bit_rotation_h(index2,x,nqubits);
      rotR[index]=rotR[index]+cospi((2.*k*i)/x)*R[index2]-sinpi((2.*k*i)/x)*I[index2];
      rotI[index]=rotI[index]+sinpi((2.*k*i)/x)*R[index2]+cospi((2.*k*i)/x)*I[index2];
    }
    rotR[index]=rotR[index];
    rotI[index]=rotI[index];
    index +=blockDim.x*gridDim.x;
  }
  
}


__global__ void vertical_proyector(double R[],double I[],double rotR[],double rotI[],int x,int nqubits,int l,int k) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  int y=nqubits/x;
  while (index<l) {
    index2=index;
    rotR[index]=R[index];
    rotI[index]=I[index];
    for(int i=1;i<y;i++) {
      index2=bit_rotation_v(index2,x,nqubits);
      rotR[index]=rotR[index]+cospi((2.*k*i)/y)*R[index2]-sinpi((2.*k*i)/y)*I[index2];
      rotI[index]=rotI[index]+sinpi((2.*k*i)/y)*R[index2]+cospi((2.*k*i)/y)*I[index2];
    }
    rotR[index]=rotR[index];
    rotI[index]=rotI[index];
    index +=blockDim.x*gridDim.x;
  }
  
}

__global__ void both_proyector(double R[],double I[],double rotR[],double rotI[],int x,int nqubits,int l,int kx,int ky) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  int y=nqubits/x;
  double cx,cy,sx,sy;
  while (index<l) {
    index2=index;
    for(int i=1;i<=x;i++) {
      index2=bit_rotation_h(index2,x,nqubits);
      cx=cospi((2.*kx*i)/double(x));
      sx=sinpi((2.*kx*i)/double(x));
      for(int j=1;j<=y;j++) {
	index2=bit_rotation_v(index2,x,nqubits);
	cy=cospi((2.*ky*j)/double(y));
	sy=sinpi((2.*ky*j)/double(y));
	rotR[index]+=(cx*cy-sx*sy)*R[index2]-(cx*sy+cy*sx)*I[index2];
	rotI[index]+=(cx*sy+cy*sx)*R[index2]+(cx*cy-sx*sy)*I[index2];
      }
    }
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void special_both_proyector(double R[],double I[],int x,int nqubits,int l,int kx,int ky,int S) {
  long long int index2;
  int y=nqubits/x;
  double cx,cy,sx,sy;
  index2=S;
  int norm=0;
  for(int i=1;i<=x;i++) {
    index2=bit_rotation_h(index2,x,nqubits);
    cx=cospi((2.*kx*i)/double(x));
    sx=sinpi((2.*kx*i)/double(x));
    for(int j=1;j<=y;j++) {
      index2=bit_rotation_v(index2,x,nqubits);
      cy=cospi((2.*ky*j)/double(y));
      sy=sinpi((2.*ky*j)/double(y));
      R[index2]+=(cx*cy-sx*sy);
      I[index2]+=(cx*sy+cy*sx);
      if (index2==S) {
	norm++;
      }
    }
  }
  normed=1./sqrt(double(nqubits*norm));
}

__global__ void special_chain_proyector(double R[],double I[],int nqubits,int l,int kx,int S) {
  long long int index2;
  double cx,sx;
  index2=S;
  int norm=0;
  for(int i=1;i<=nqubits;i++) {
    index2=bit_rotation_h(index2,nqubits,nqubits);
    cx=cospi((2.*kx*i)/double(nqubits));
    sx=sinpi((2.*kx*i)/double(nqubits));
    R[index2]+=cx;
    I[index2]+=sx;
    if (index2==S) {
      norm++;
    }
  }
  
  normed=1./sqrt(double(nqubits*norm));
}

__global__ void reflection_proyector(double R[],double I[],int nqubits,int lambda,int S) {
  long long int index2;
  index2=S;
  int norm=1;
  R[index2]=1;
  index2=bit_reflection_h(index2,nqubits);
  R[index2]=lambda*1;
  if (index2==S) {
    norm++;
  }
  normed=1./sqrt(2./double(norm));
}

__global__ void special_both_proyector(double R[],double I[],int x,int nqubits,int l,int kx,int ky,int S,double sr,double si) {
  //checar el signo de la parte imaginaria
  long long int index2;
  int y=nqubits/x;
  double cx,cy,sx,sy;
  index2=S;
  int norm=0;
  for(int i=1;i<=x;i++) {
    index2=bit_rotation_h(index2,x,nqubits);
    cx=cospi((2.*kx*i)/double(x));
    sx=sinpi((2.*kx*i)/double(x));
    for(int j=1;j<=y;j++) {
      index2=bit_rotation_v(index2,x,nqubits);
      cy=cospi((2.*ky*j)/double(y));
      sy=sinpi((2.*ky*j)/double(y));
      R[index2]+=(cx*cy-sx*sy)*sr-(cx*sy+cy*sx)*si;
      I[index2]+=(cx*sy+cy*sx)*sr+(cx*cy-sx*sy)*si;
      if (index2==S) {
	norm++;
      }
    }
  }
  normed=1./sqrt(double(norm));
}


__global__ void proyected_dot(double R[],double I[],double rotR[],double rotI[],int x,int nqubits,int l,int kx,int ky, int S[]) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  //   long long int index2,norm;
  int y=nqubits/x;
  //   int i,j;
  while (index<l) {
    double cx,cy,sx,sy;
    index2=S[index];
    int norm=0;
    for(int i=1;i<=x;i++) {
      index2=bit_rotation_h(index2,x,nqubits);
      cx=cospi((2.*kx*i)/double(x));
      sx=sinpi((2.*kx*i)/double(x));
      for(int j=1;j<=y;j++) {
	index2=bit_rotation_v(index2,x,nqubits);
	cy=cospi((2.*ky*j)/double(y));
	sy=sinpi((2.*ky*j)/double(y));
	rotR[index]=rotR[index]+(cx*cy-sx*sy)*R[index2]-(cx*sy+cy*sx)*I[index2];
	rotI[index]=rotI[index]+(cx*sy+cy*sx)*R[index2]+(cx*cy-sx*sy)*I[index2];
	if (index2==S[index]) {
	  norm++;
	}
      }
    }
    rotR[index]=rotR[index]/sqrt(double(nqubits*norm));
    rotI[index]=rotI[index]/sqrt(double(nqubits*norm));
    index +=blockDim.x*gridDim.x;
  }
  
}


__global__ void proyected_dot_chain(double R[],double I[],double rotR[],double rotI[],int nqubits,int l,int kx,int S[]) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  //   long long int index2,norm;
  
  //   int i,j;
  while (index<l) {
    double cx,sx;
    index2=S[index];
    int norm=0;
    for(int i=1;i<=nqubits;i++) {
      index2=bit_rotation_h(index2,nqubits,nqubits);
      cx=cospi((2.*kx*i)/double(nqubits));
      sx=sinpi((2.*kx*i)/double(nqubits));
      rotR[index]=rotR[index]+(cx)*R[index2]-(sx)*I[index2];
      rotI[index]=rotI[index]+(sx)*R[index2]+(cx)*I[index2];
      if (index2==S[index]) {
	norm++;
      }
      
    }
    rotR[index]=rotR[index]/sqrt(double(nqubits*norm));
    rotI[index]=rotI[index]/sqrt(double(nqubits*norm));
    index +=blockDim.x*gridDim.x;
  }
  
}

__global__ void proyected_dot_reflection(double R[],double I[],double rotR[],double rotI[],int nqubits,int l,int lambda,int S[]) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int index2;
  //   long long int index2,norm;
  
  //   int i,j;
  while (index<l) {
    index2=S[index];
    int norm=1;
    rotR[index]+= R[index2];
    rotI[index]+= I[index2];
    index2=bit_reflection_h(index2,nqubits);
    rotR[index]+= lambda*R[index2];
    rotI[index]+= lambda*I[index2];
    if (index2==S[index]) {
      norm++;
    }   
    rotR[index]=rotR[index]/sqrt(2./double(norm));
    rotI[index]=rotI[index]/sqrt(2./double(norm));
    index +=blockDim.x*gridDim.x;
  } 
}


__global__ void dev_sum(int l,double R1[],double I1[],double R2[], double I2[]) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  while (index<l) {
    R1[index]+=R2[index];
    I1[index]+=I2[index];
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void Ui_kernel(int n,int m,double R[],double I[],double mcos,double msin,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int in=1<<n;
  long long int im=1<<m;
  while (index<l) {
    Complex i(0,1);
    Complex a=Complex(R[index],I[index])*mcos;
    int sigz=(((index&in)/in)+((index&im)/im))%2;
    Complex b=Complex(R[index],I[index])*ipow(-1,sigz)*msin;
    b=b*i;
    R[index]=(a-b).real;
    I[index]=(a-b).imag;
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void Uk_kernel(int k,double R[],double I[],double bx,double by,double bz,double mcos,double msin,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2=1<<k;
  Complex i(0,1);
  while (index<l) {
    //bitwise aqui
    //no tiene a i2
    if ((index&i2)==0) {
      Complex a=Complex(R[index],I[index])*mcos;
      Complex b=Complex(R[index],I[index])*bz;
      Complex c=Complex(R[index+i2],I[index+i2])*bx;
      Complex d=Complex(R[index+i2],I[index+i2])*by*-1;
      d=d*i;
      b=b+c+d;
      b=(b*i)*msin;
      Complex a2=Complex(R[index+i2],I[index+i2])*mcos;
      Complex b2=Complex(R[index+i2],I[index+i2])*bz*-1;
      Complex c2=Complex(R[index],I[index])*bx;
      Complex d2=Complex(R[index],I[index])*by;
      d2=d2*i;
      b2=b2+c2+d2;
      b2=(b2*i)*msin;
      R[index]=(a-b).real;
      I[index]=(a-b).imag;
      R[index+i2]=(a2-b2).real;
      I[index+i2]=(a2-b2).imag;
    }
    index+=blockDim.x*gridDim.x;
  }
}

__global__ void sigma_x(double R[],double I[],double sumdxR[],double sumdxI[],int i,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2;
  while (index<l) {
    i2=ipow(2,i);
    sumdxR[index]=R[index+i2*ipow(-1,(index&i2)/i2)];
    sumdxI[index]=I[index+i2*ipow(-1,(index&i2)/i2)];
    
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void sigma_xsigma_y(double R[],double I[],double sumdxR[],double sumdxI[],int i,int j,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2,i3;
  while (index<l) {
    i2=ipow(2,i);
    i3=ipow(2,j);
    sumdxR[index]=R[index+i2*ipow(-1,(index&i2)/i2)]+ipow(-1,(index&i3)/i3+2)*I[index+i3*ipow(-1,(index&i3)/i3)];
    sumdxI[index]=I[index+i2*ipow(-1,(index&i2)/i2)]+ipow(-1,(index&i3)/i3+1)*R[index+i3*ipow(-1,(index&i3)/i3)];
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void sumsigma_x(double R[],double I[],double sumdxR[],double sumdxI[],int nqubits,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2;
  while (index<l) {
    int i=0;
    sumdxR[index]=0.;
    sumdxI[index]=0.;
    while (i<nqubits) {
      i2=ipow(2,i);
      sumdxR[index]+=R[index+i2*ipow(-1,(index&i2)/i2)];
      sumdxI[index]+=I[index+i2*ipow(-1,(index&i2)/i2)];
      i++;
    }
    index +=blockDim.x*gridDim.x;
  }
}


__global__ void sumsigma_z(double R[],double I[],double sumdzR[],double sumdzI[],int nqubits,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2;
  while (index<l) {
    int i=0;
    sumdzR[index]=0.;
    sumdzI[index]=0.;
    while (i<nqubits) {
      i2=ipow(2,i);
      sumdzR[index]+=ipow(-1,(index&i2)/i2);
      sumdzI[index]+=ipow(-1,(index&i2)/i2);
      i++;
    }
    sumdzR[index]*=R[index];
    sumdzI[index]*=I[index];
    index +=blockDim.x*gridDim.x;
  }
}


__global__ void sumsigma_y(double R[],double I[],double sumdyR[],double sumdyI[],int nqubits,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2;
  while (index<l) {
    int i=0;
    sumdyR[index]=0.;
    sumdyI[index]=0.;
    while (i<nqubits) {
      i2=ipow(2,i);
      sumdyR[index]+=ipow(-1,(index&i2)/i2+2)*I[index+i2*ipow(-1,(index&i2)/i2)];
      sumdyI[index]+=ipow(-1,(index&i2)/i2+1)*R[index+i2*ipow(-1,(index&i2)/i2)];
      i++;
    }
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void prodsigma_x(double R[],double I[],double sumdxR[],double sumdxI[],int nqubits,int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  long long int i2;
  while (index<l) {
    int i=0;
    sumdxR[index]=1.;
    sumdxI[index]=1.;
    while (i<nqubits) {
      i2=ipow(2,i);
      sumdxR[index]=(R[index+i2*ipow(-1,(index/i2)%2)]*sumdxR[index])-(I[index+i2*ipow(-1,(index/i2)%2)]*sumdxI[index]);
      sumdxI[index]=(I[index+i2*ipow(-1,(index/i2)%2)]*sumdxR[index])+(R[index+i2*ipow(-1,(index/i2)%2)]*sumdxI[index]);
      i++;
    }
    index +=blockDim.x*gridDim.x;
  }
}



__device__ void matomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull =
  (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
//   do {
//     assumed = old;
//     old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +__longlong_as_double(assumed)));
//   } while (assumed != old);
  // return __longlong_as_double(old);
}



__global__ void QFT(int k,double R[],double I[],int l,int nqubits) {
  int index=threadIdx.x + blockIdx.x * blockDim.x;
  while (((index/__float2int_rz(__powf(2,k)))%2==0) && (index<l)) {
    int i2=__float2int_rz(__powf(2,k));
    double x=R[index];
    double y=I[index];
    R[index]=(R[index]+R[index+i2])*(1/sqrtf(2));
    I[index]=(I[index]+I[index+i2])*(1/sqrtf(2));
    R[index+i2]=(x-R[index+i2])*(1/sqrtf(2));
    I[index+i2]=(y-I[index+i2])*(1/sqrtf(2));
    for(int i=k+1;i<nqubits;i++) {
      if((index/__float2int_rz(__powf(2,i)))%2==1) {
	Complex a=Complex(R[index+i2],I[index+i2]);
	Complex b=Complex(cosf(2*3.141592654f/__powf(2,i)),sinf(2*3.141592654f/__powf(2,i)));
	a=a*b;
	R[index+i2]=a.real;
	I[index+i2]=a.imag;
      }
    }
    index +=blockDim.x*gridDim.x;
  }
}




__global__ void density_matrix(int which,int ndim,int tarjet1,int tarjet2,double R[],double I[],double densR[],double densI[],int l) {
  int index=threadIdx.x+blockIdx.x*blockDim.x;
  __shared__ Complex cache[512];
  Complex temp=Complex(0,0);
  int cacheindex=threadIdx.x;
  int i1,i2;
  
  while (index<l) {
    i1=trans(index,which,tarjet1);
    i2=trans(index,which,tarjet2);
    Complex a=Complex(R[i1],I[i1]);
    Complex b=Complex(R[i2],(-1)*I[i2]);
    a=(a*b);
    temp=temp+a;
    index+=blockDim.x*gridDim.x;
  }
  cache[cacheindex]=temp;
  __syncthreads();
  
  
  int n = blockDim.x/2;
  while (n != 0) {
    if (cacheindex < n) {
      cache[cacheindex] = cache[cacheindex] + cache[cacheindex + n];
    }
    __syncthreads();
    n /= 2;
  }
  if (cacheindex==0) {
    matomicAdd(&densR[(tarjet1*ndim)+tarjet2],cache[0].real);
    matomicAdd(&densI[(tarjet1*ndim)+tarjet2],cache[0].imag);
  }
}

//revisar shared cache  
__global__ void timed_dot(int time,double AR[],double AI[],double BR[],double BI[],double resR[],double resI[],int l) {
  int index=threadIdx.x+blockIdx.x*blockDim.x;
  //__shared__ Complex cache;
  Complex temp=Complex(0,0);
  int cacheindex=threadIdx.x;
  
  while (index<l) {
    Complex a=Complex(AR[index],AI[index]);
    Complex b=Complex(BR[index],(-1)*BI[index]);
    a=(a*b);
    temp=temp+a;
    index+=blockDim.x*gridDim.x;
  }
  //cache=temp;
  __syncthreads();
  
  
  int n = blockDim.x/2;
  while (n != 0) {
    if (cacheindex < n) {
      //cache[cacheindex] = cache[cacheindex] + cache[cacheindex + n];
    }
    __syncthreads();
    n /= 2;
  }
  //   if (cacheindex==0) {
  //     matomicAdd(&resR[time],cache.real);  
  //     matomicAdd(&resI[time],cache.imag);
  //   }
}

__global__ void to_zero(double A[],double B[],int l) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  while (index<l) {
    A[index]=0.;
    B[index]=0.;
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void index_one(double A[],double B[],int l,int in) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  while (index<l) {
    A[index]=0.;
    B[index]=0.;
    if(index==in) {
      A[index]=1.;
    }
    index +=blockDim.x*gridDim.x;
  }
}



__global__ void devcpy(int l,double A1[],double A2[],double B1[],double B2[]) {
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  while (index<l) {
    B1[index]=A1[index];
    B2[index]=A2[index];
    index +=blockDim.x*gridDim.x;
  }
}

__global__ void times_norm(double A[],double B[],int l) {
  //VER SI AQUI VA LA CONGUDADA
  long long int index=threadIdx.x + blockIdx.x * blockDim.x;
  while (index<l) {
    A[index]=A[index]*normed;
    B[index]=B[index]*normed;
    index +=blockDim.x*gridDim.x;
  }
}



#endif
